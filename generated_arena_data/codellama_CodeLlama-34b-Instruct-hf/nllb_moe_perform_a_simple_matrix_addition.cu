
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for matrix addition using bfloat16
__global__ void matrix_addition_kernel_bf16(const float* matrix1, const float* matrix2, float* output, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        __hip_bfloat16 a = float_to_bfloat16(matrix1[row * n + col]);
        __hip_bfloat16 b = float_to_bfloat16(matrix2[row * n + col]);
        output[row * n + col] = bfloat16_to_float(__hadd(a, b));
    }
}

extern "C" {

void simple_matrix_addition(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = matrix1_dim0;
    int input_dim = matrix1_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_matrix2, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * input_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((input_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_addition_kernel_bf16<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, batch_size, input_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * input_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"