
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for matrix subtraction using bfloat16
__global__ void matmul_sub_kernel_bf16(const float* matrix1, const float* matrix2, float* output, 
                                       int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            __hip_bfloat16 a = float_to_bfloat16(matrix1[row * k + i]);
            __hip_bfloat16 b = float_to_bfloat16(matrix2[col * k + i]);
            sum += bfloat16_to_float(__hsub(a, b));
        }
        output[row * n + col] = sum;
    }
}

extern "C" {

void simple_matrix_subtraction(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input matrices
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output matrix (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = matrix1_dim0;
    int input_dim = matrix1_dim1;
    int output_dim = matrix2_dim0;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_matrix2, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_sub_kernel_bf16<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, batch_size, output_dim, input_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"