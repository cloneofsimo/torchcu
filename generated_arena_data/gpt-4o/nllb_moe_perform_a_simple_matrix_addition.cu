
#include <hip/hip_runtime.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// CUDA kernel for matrix addition
__global__ void matrix_addition_kernel(const float* matrix1, const float* matrix2, float* output, int num_rows, int num_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows && col < num_cols) {
        int index = row * num_cols + col;
        output[index] = matrix1[index] + matrix2[index];
    }
}

extern "C" {

void simple_matrix_addition(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract matrix1
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    // Extract matrix2
    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output matrix (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int num_rows = matrix1_dim0;
    int num_cols = matrix1_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, num_rows * num_cols * sizeof(float));
    hipMalloc(&d_matrix2, num_rows * num_cols * sizeof(float));
    hipMalloc(&d_output, num_rows * num_cols * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, num_rows * num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, num_rows * num_cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((num_cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (num_rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_addition_kernel<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, num_rows, num_cols
    );

    // Copy result back to host
    hipMemcpy(output, d_output, num_rows * num_cols * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"
