
#include <hip/hip_runtime.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// CUDA kernel for simple matrix subtraction
__global__ void matrix_subtraction_kernel(const float* matrix1, const float* matrix2, float* output, 
                                          int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        output[row * n + col] = matrix1[row * n + col] - matrix2[row * n + col];
    }
}

extern "C" {

void simple_matrix_subtraction(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract first matrix
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    // Extract second matrix
    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int m = matrix1_dim0;
    int n = matrix1_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, m * n * sizeof(float));
    hipMalloc(&d_matrix2, m * n * sizeof(float));
    hipMalloc(&d_output, m * n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_subtraction_kernel<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, m, n
    );

    // Copy result back to host
    hipMemcpy(output, d_output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"
