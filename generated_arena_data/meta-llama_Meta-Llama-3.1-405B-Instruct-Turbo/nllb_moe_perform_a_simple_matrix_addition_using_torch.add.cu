
#include <hip/hip_runtime.h>

#include <stdarg.h>

__global__ void matrix_addition_kernel(const float* matrix1, const float* matrix2, float* output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        output[idx] = matrix1[idx] + matrix2[idx];
    }
}

extern "C" {

void simple_matrix_addition(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract matrix1
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    // Extract matrix2
    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output matrix (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int rows = matrix1_dim0;
    int cols = matrix1_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, rows * cols * sizeof(float));
    hipMalloc(&d_matrix2, rows * cols * sizeof(float));
    hipMalloc(&d_output, rows * cols * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_addition_kernel<<<numBlocks, threadsPerBlock>>>(d_matrix1, d_matrix2, d_output, rows, cols);

    // Copy result back to host
    hipMemcpy(output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"
