
#include <hip/hip_runtime.h>

#include <stdarg.h>

// CUDA kernel for matrix multiplication
__global__ void matrix_multiply_kernel(const float* matrix1, const float* matrix2, float* output, 
                                        int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += matrix1[row * k + i] * matrix2[i * n + col];
        }
        output[row * n + col] = sum;
    }
}

extern "C" {

void simple_matrix_multiplication(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int m = matrix1_dim0;
    int k = matrix1_dim1;
    int n = matrix2_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, m * k * sizeof(float));
    hipMalloc(&d_matrix2, k * n * sizeof(float));
    hipMalloc(&d_output, m * n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, k * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_multiply_kernel<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, m, n, k
    );

    // Copy result back to host
    hipMemcpy(output, d_output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"
