#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

// CUDA kernel for matrix subtraction
__global__ void matrix_subtraction_kernel(const float* matrix1, const float* matrix2, float* output, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        output[row * n + col] = matrix1[row * n + col] - matrix2[row * n + col];
    }
}

extern "C" {

void simple_matrix_subtraction(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input matrix1
    const float* matrix1 = va_arg(args, const float*);
    int matrix1_dim0 = va_arg(args, int);
    int matrix1_dim1 = va_arg(args, int);

    // Extract input matrix2
    const float* matrix2 = va_arg(args, const float*);
    int matrix2_dim0 = va_arg(args, int);
    int matrix2_dim1 = va_arg(args, int);

    // Extract output matrix (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int rows = matrix1_dim0;
    int cols = matrix1_dim1;

    // Allocate device memory
    float *d_matrix1, *d_matrix2, *d_output;
    hipMalloc(&d_matrix1, rows * cols * sizeof(float));
    hipMalloc(&d_matrix2, rows * cols * sizeof(float));
    hipMalloc(&d_output, rows * cols * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_matrix1, matrix1, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_subtraction_kernel<<<numBlocks, threadsPerBlock>>>(
        d_matrix1, d_matrix2, d_output, rows, cols
    );

    // Copy result back to host
    hipMemcpy(output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_output);
}

}  // extern "C"
