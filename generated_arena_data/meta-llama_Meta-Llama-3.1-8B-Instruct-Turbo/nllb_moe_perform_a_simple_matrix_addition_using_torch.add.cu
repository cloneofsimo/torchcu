
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for matrix transpose using bfloat16
__global__ void transpose_kernel_bf16(const float* input_matrix, float* output_matrix, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        output_matrix[col * m + row] = input_matrix[row * n + col];
    }
}

// CUDA kernel for matrix transpose using bfloat16
__global__ void transpose_kernel_bf16_2(const float* input_matrix, float* output_matrix, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        output_matrix[row * n + col] = input_matrix[col * m + row];
    }
}

extern "C" {

void matrix_transpose(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_matrix = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output_matrix = va_arg(args, float*);

    va_end(args);

    int batch_size = input_dim0;
    int input_dim = input_dim1;

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * input_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_matrix, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((input_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    transpose_kernel_bf16<<<numBlocks, threadsPerBlock>>>(
        d_input, d_output, batch_size, input_dim
    );

    // Copy result back to host
    hipMemcpy(output_matrix, d_output, batch_size * input_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

void matrix_transpose_2(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_matrix = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output_matrix = va_arg(args, float*);

    va_end(args);

    int batch_size = input_dim0;
    int input_dim = input_dim1;

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * input_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_matrix, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((input_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    transpose_kernel_bf16_2<<<numBlocks, threadsPerBlock>>>(
        d_input, d_output, batch_size, input_dim
    );

    // Copy result back to host
    hipMemcpy(output_matrix, d_output, batch_size * input_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
