
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for MSE loss calculation using bfloat16
__global__ void mse_loss_kernel_bf16(const float* input_tensor, const float* target_tensor, float* loss, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        __hip_bfloat16 a = float_to_bfloat16(input_tensor[idx]);
        __hip_bfloat16 b = float_to_bfloat16(target_tensor[idx]);
        __hip_bfloat16 diff = a - b;
        __hip_bfloat16 squared_diff = diff * diff;
        atomicAdd(loss, bfloat16_to_float(squared_diff));
    }
}

extern "C" {

void torch_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_size = va_arg(args, int);

    const float* target_tensor = va_arg(args, const float*);
    int target_tensor_size = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* loss = va_arg(args, float*);

    va_end(args);

    // Allocate device memory
    float *d_input, *d_target, *d_loss;
    hipMalloc(&d_input, input_tensor_size * sizeof(float));
    hipMalloc(&d_target, target_tensor_size * sizeof(float));
    hipMalloc(&d_loss, sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, input_tensor, input_tensor_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target_tensor, target_tensor_size * sizeof(float), hipMemcpyHostToDevice);

    // Initialize loss on device
    hipMemset(d_loss, 0, sizeof(float));

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((input_tensor_size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    mse_loss_kernel_bf16<<<numBlocks, threadsPerBlock>>>(d_input, d_target, d_loss, input_tensor_size);

    // Copy result back to host
    hipMemcpy(loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    // Calculate final loss
    loss[0] /= input_tensor_size;

    // Free device memory
    hipFree(d_input);
    hipFree(d_target);
    hipFree(d_loss);
}

}  // extern "C"
