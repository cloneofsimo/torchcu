#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end
#include <hipDNN.h>

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

extern "C" {

void learned_positional_encoding_bf16_max_filter(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract learned positional encoding tensor
    const float* learned_positional_encoding = va_arg(args, const float*);
    int learned_positional_encoding_dim0 = va_arg(args, int);
    int learned_positional_encoding_dim1 = va_arg(args, int);
    int learned_positional_encoding_dim2 = va_arg(args, int);
    int learned_positional_encoding_dim3 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // CUDA context creation and error checking
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // Allocate device memory
    float *d_input, *d_learned_positional_encoding, *d_output;
    hipMalloc(&d_input, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));
    hipMalloc(&d_learned_positional_encoding, learned_positional_encoding_dim0 * learned_positional_encoding_dim1 * learned_positional_encoding_dim2 * learned_positional_encoding_dim3 * sizeof(float));
    hipMalloc(&d_output, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_learned_positional_encoding, learned_positional_encoding, learned_positional_encoding_dim0 * learned_positional_encoding_dim1 * learned_positional_encoding_dim2 * learned_positional_encoding_dim3 * sizeof(float), hipMemcpyHostToDevice);

    // Create cudnn tensor descriptors
    hipdnnTensorDescriptor_t input_tensor_desc, learned_positional_encoding_desc, output_desc;
    hipdnnCreateTensorDescriptor(&input_tensor_desc);
    hipdnnCreateTensorDescriptor(&learned_positional_encoding_desc);
    hipdnnCreateTensorDescriptor(&output_desc);

    // Set tensor descriptor dimensions (NHWC)
    hipdnnSetTensor4dDescriptor(input_tensor_desc, HIPDNN_DATA_FLOAT, 1, input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);
    hipdnnSetTensor4dDescriptor(learned_positional_encoding_desc, HIPDNN_DATA_FLOAT, 1, learned_positional_encoding_dim0, learned_positional_encoding_dim1, learned_positional_encoding_dim2, learned_positional_encoding_dim3);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_DATA_FLOAT, 1, input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);

    // Create cudnn filter descriptor for max pooling
    hipdnnFilterDescriptor_t max_pool_filter_desc;
    hipdnnCreateFilterDescriptor(&max_pool_filter_desc);
    hipdnnSetFilter4dDescriptor(max_pool_filter_desc, HIPDNN_DATA_FLOAT, CUDNN_TENSOR_FORMAT_NHWC, 3, 3, 1, 1);

    // Create cudnn pooling descriptor
    hipdnnPoolingDescriptor_t max_pool_desc;
    hipdnnCreatePoolingDescriptor(&max_pool_desc);
    hipdnnSetPoolingNdDescriptor(max_pool_desc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, 3, 3, 1, 1, 1, 1, 1, 1);

    // Perform the learned positional encoding addition on the device
    hipdnnAddTensor(cudnnHandle, CUDNN_ADD_SAME_C, 
                  &input_tensor_desc, d_input,
                  &learned_positional_encoding_desc, d_learned_positional_encoding,
                  &output_desc, d_output);

    // Perform max pooling with bfloat16 conversion using cudnn
    hipdnnDataType_t data_type = CUDNN_DATA_BFLOAT16;
    cudnnSetTensorDescriptorEx(input_tensor_desc, data_type, 1, input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);
    cudnnSetTensorDescriptorEx(output_desc, data_type, 1, input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);

    hipdnnPoolingForward(cudnnHandle, max_pool_desc, 
                        input_tensor_desc, d_output,
                        output_desc, d_output);
                        
    // Convert back to float32
    cudnnSetTensorDescriptorEx(output_desc, HIPDNN_DATA_FLOAT, 1, input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);

    // Copy result back to host
    hipMemcpy(output, d_output, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory and cudnn descriptors
    hipFree(d_input);
    hipFree(d_learned_positional_encoding);
    hipFree(d_output);
    hipdnnDestroyTensorDescriptor(input_tensor_desc);
    hipdnnDestroyTensorDescriptor(learned_positional_encoding_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(max_pool_filter_desc);
    hipdnnDestroyPoolingDescriptor(max_pool_desc);
    hipdnnDestroy(cudnnHandle);
}

}  // extern "C"
