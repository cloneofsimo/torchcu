#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cuda_math.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include <algorithm>
#include <cmath>
#include <complex>
#include <iostream>

// Helper function to convert float to half
__device__ __forceinline__ half float_to_half(float f) {
    return __float2half_rn(f);
}

// Helper function to convert half to float
__device__ __forceinline__ float half_to_float(half h) {
    return __half2float(h);
}

// CUDA kernel for fading-in with exponential function
__global__ void fading_in_kernel(const float* input, float* output, float alpha, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    output[i] = (1 - expf(-alpha * i)) * input[i];
  }
}

// CUDA kernel for linear layer (using CUDNN)
__global__ void linear_layer_kernel(const float* input, const float* weight, float* output, int batch_size, int in_features,
                                      int out_features) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < batch_size) {
    float sum = 0.0f;
    for (int j = 0; j < in_features; ++j) {
      sum += input[i * in_features + j] * weight[j * out_features + threadIdx.y];
    }
    output[i * out_features + threadIdx.y] = fmaxf(sum, 0.0f); // ReLU activation
  }
}

extern "C" {

void torch_function(int num_args, ...) {
  va_list args;
  va_start(args, num_args);

  // Extract input tensors
  const float* input_tensor = va_arg(args, const float*);
  int input_tensor_dim0 = va_arg(args, int);
  int input_tensor_dim1 = va_arg(args, int);

  const float* weight = va_arg(args, const float*);
  int weight_dim0 = va_arg(args, int);
  int weight_dim1 = va_arg(args, int);

  const float* alpha = va_arg(args, const float*);

  // Extract output tensor (assuming it's preallocated)
  float* output = va_arg(args, float*);

  va_end(args);

  int batch_size = input_tensor_dim0;
  int input_dim = input_tensor_dim1;

  // Allocate device memory
  float *d_input, *d_weight, *d_output, *d_alpha;
  hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
  hipMalloc(&d_weight, weight_dim0 * weight_dim1 * sizeof(float));
  hipMalloc(&d_output, batch_size * input_dim * sizeof(float));
  hipMalloc(&d_alpha, sizeof(float));

  // Copy input data to device
  hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);

  // CUDNN FFT
  hipdnnHandle_t cudnnHandle;
  hipdnnCreate(&cudnnHandle);

  hipdnnTensorDescriptor_t inputDesc, outputDesc;
  hipdnnCreateTensorDescriptor(&inputDesc);
  hipdnnCreateTensorDescriptor(&outputDesc);

  hipdnnDataType_t dataType = HIPDNN_DATA_FLOAT;
  int dimA[4] = {1, batch_size, 1, input_dim};
  hipdnnSetTensorNdDescriptor(inputDesc, dataType, 4, dimA, nullptr);
  hipdnnSetTensorNdDescriptor(outputDesc, dataType, 4, dimA, nullptr);

  cudnnPlan_t plan;
  cudnnCreatePlan(&plan);
  cudnnSetPlanForward(plan, inputDesc, outputDesc, CUDNN_FFT_FORWARD, dataType, CUDNN_FFT_DEFAULT, CUDNN_FFT_DEFAULT, 0);

  cudnnExecutePlan(plan, cudnnHandle, d_input, d_output);

  // CUDNN Linear Layer
  hipdnnTensorDescriptor_t input_linear_desc, output_linear_desc, weight_linear_desc;
  hipdnnCreateTensorDescriptor(&input_linear_desc);
  hipdnnCreateTensorDescriptor(&output_linear_desc);
  hipdnnCreateTensorDescriptor(&weight_linear_desc);

  int dimB[4] = {1, batch_size, 1, weight_dim1};
  int dimC[4] = {1, 1, 1, weight_dim0 * weight_dim1};
  hipdnnSetTensorNdDescriptor(input_linear_desc, dataType, 4, dimB, nullptr);
  hipdnnSetTensorNdDescriptor(output_linear_desc, dataType, 4, dimB, nullptr);
  hipdnnSetTensorNdDescriptor(weight_linear_desc, dataType, 4, dimC, nullptr);

  hipdnnFilterDescriptor_t filterDesc;
  hipdnnCreateFilterDescriptor(&filterDesc);
  hipdnnSetFilterNdDescriptor(filterDesc, dataType, HIPDNN_TENSOR_NCHW, 4, dimC, nullptr);

  hipdnnConvolutionDescriptor_t convDesc;
  hipdnnCreateConvolutionDescriptor(&convDesc);

  hipdnnSetConvolutionNdDescriptor(convDesc, 0, CUDNN_CROSS_CHANNEL_DIVISION, 0, 0, 1, 1, 1, 1, 1, 1, 1, HIPDNN_DATA_FLOAT);
  hipdnnConvolutionForward(cudnnHandle, 1.0f, input_linear_desc, d_output, filterDesc, d_weight, convDesc,
                          HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, nullptr, 0.0f, output_linear_desc, d_output);

  // Fading-in
  fading_in_kernel<<<(input_dim + 255) / 256, 256>>>(d_output, d_output, *d_alpha, input_dim);

  // Inverse CUDNN FFT
  cudnnSetPlanForward(plan, inputDesc, outputDesc, CUDNN_FFT_INVERSE, dataType, CUDNN_FFT_DEFAULT, CUDNN_FFT_DEFAULT, 0);
  cudnnExecutePlan(plan, cudnnHandle, d_output, d_output);

  // Copy result back to host
  hipMemcpy(output, d_output, batch_size * input_dim * sizeof(float), hipMemcpyDeviceToHost);

  // Free CUDNN resources
  cudnnDestroyPlan(plan);
  hipdnnDestroyTensorDescriptor(inputDesc);
  hipdnnDestroyTensorDescriptor(outputDesc);
  hipdnnDestroyTensorDescriptor(input_linear_desc);
  hipdnnDestroyTensorDescriptor(output_linear_desc);
  hipdnnDestroyTensorDescriptor(weight_linear_desc);
  hipdnnDestroyFilterDescriptor(filterDesc);
  hipdnnDestroyConvolutionDescriptor(convDesc);
  hipdnnDestroy(cudnnHandle);

  // Free device memory
  hipFree(d_input);
  hipFree(d_weight);
  hipFree(d_output);
  hipFree(d_alpha);
}

}  // extern "C"
