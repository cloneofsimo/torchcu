#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <math.h> 
#include <hipDNN.h>

#define CHECK(x)                                              \
  do                                                          \
  {                                                           \
    hipdnnStatus_t status = (x);                               \
    if (status != HIPDNN_STATUS_SUCCESS)                       \
    {                                                           \
      fprintf(stderr, "CUDNN error: %s:%d: %s\n", __FILE__,    \
              __LINE__, hipdnnGetErrorString(status));          \
      exit(EXIT_FAILURE);                                     \
    }                                                           \
  } while (0)

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

extern "C" {

void torch_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);
    int weight_dim2 = va_arg(args, int);
    int weight_dim3 = va_arg(args, int);

    const float* bias = va_arg(args, const float*);
    int bias_dim = va_arg(args, int);

    const int* target_tensor = va_arg(args, const int*);
    int target_tensor_dim0 = va_arg(args, int);
    int target_tensor_dim1 = va_arg(args, int);
    int target_tensor_dim2 = va_arg(args, int);
    int target_tensor_dim3 = va_arg(args, int);

    const float* sparsity_weight_ptr = va_arg(args, const float*);
    float sparsity_weight = *sparsity_weight_ptr;

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // CUDA Initialization
    hipdnnHandle_t cudnn_handle;
    CHECK(hipdnnCreate(&cudnn_handle));

    // Define tensor descriptors
    hipdnnTensorDescriptor_t input_desc, weight_desc, output_desc, target_desc;
    CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CHECK(hipdnnCreateTensorDescriptor(&weight_desc));
    CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CHECK(hipdnnCreateTensorDescriptor(&target_desc));

    // Set tensor descriptors
    CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_DATA_FLOAT, 1, input_tensor_dim0, 
                                     input_tensor_dim1, input_tensor_dim2, input_tensor_dim3));
    CHECK(hipdnnSetTensor4dDescriptor(weight_desc, HIPDNN_DATA_FLOAT, 1, weight_dim0,
                                     weight_dim1, weight_dim2, weight_dim3));
    CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_DATA_FLOAT, 1, target_tensor_dim0,
                                     target_tensor_dim1, target_tensor_dim2, target_tensor_dim3));
    CHECK(hipdnnSetTensor4dDescriptor(target_desc, HIPDNN_DATA_INT32, 1, target_tensor_dim0,
                                     target_tensor_dim1, target_tensor_dim2, target_tensor_dim3));

    // Define convolution descriptor
    hipdnnConvolutionDescriptor_t conv_desc;
    CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, 1, 1, 1, 1, 1, 1, HIPDNN_CONVOLUTION));

    // Allocate device memory
    float *d_input, *d_weight, *d_bias, *d_output;
    int *d_target;
    hipMalloc(&d_input, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float));
    hipMalloc(&d_bias, bias_dim * sizeof(float));
    hipMalloc(&d_output, target_tensor_dim0 * target_tensor_dim1 * target_tensor_dim2 * target_tensor_dim3 * sizeof(float));
    hipMalloc(&d_target, target_tensor_dim0 * target_tensor_dim1 * target_tensor_dim2 * target_tensor_dim3 * sizeof(int));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, bias_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target_tensor, target_tensor_dim0 * target_tensor_dim1 * target_tensor_dim2 * target_tensor_dim3 * sizeof(int), hipMemcpyHostToDevice);

    // Perform convolution
    CHECK(hipdnnConvolutionForward(cudnn_handle,
                                  &alpha, conv_desc, d_input, input_desc,
                                  d_weight, weight_desc,
                                  &beta, d_output, output_desc));

    // Add bias
    CHECK(hipdnnAddTensor(cudnn_handle, 
                         &alpha, d_bias, input_desc,
                         &beta, d_output, output_desc));

    // Calculate cross-entropy loss with cudnn
    cudnnSoftmaxDescriptor_t softmax_desc;
    CHECK(cudnnCreateSoftmaxDescriptor(&softmax_desc));
    CHECK(cudnnSetSoftmaxDescriptor(softmax_desc, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE));

    float loss;
    CHECK(hipdnnSoftmaxForward(cudnn_handle, softmax_desc, d_output, output_desc, &loss));

    // Calculate KL divergence loss
    float kl_div_loss;
    hipMalloc(&kl_div_loss, sizeof(float));
    CHECK(hipLaunchKernel(
        // Kernel function
        (const void *) &kl_div_loss_kernel,
        // Threads per block
        dim3(1, 1, 1),
        // Blocks per grid
        dim3(1, 1, 1),
        // Shared memory
        0,
        // Streams
        0,
        // Kernel arguments
        d_weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3,
        &kl_div_loss,
        &sparsity_weight
    ));

    // Copy result back to host
    hipMemcpy(output, d_output, target_tensor_dim0 * target_tensor_dim1 * target_tensor_dim2 * target_tensor_dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);
    hipFree(d_target);

    // Free CUDA descriptors
    CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CHECK(hipdnnDestroyTensorDescriptor(weight_desc));
    CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CHECK(hipdnnDestroyTensorDescriptor(target_desc));
    CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CHECK(cudnnDestroySoftmaxDescriptor(softmax_desc));

    // Destroy CUDA handle
    CHECK(hipdnnDestroy(cudnn_handle));
}

__global__ void kl_div_loss_kernel(const float* weight, int weight_size, float* kl_div_loss, float sparsity_weight) {
    float sum = 0.0f;
    for (int i = 0; i < weight_size; ++i) {
        float val = weight[i];
        sum += sparsity_weight * (logf(fabsf(val) + 1e-6f) - val);
    }
    *kl_div_loss = sum;
}

}  // extern "C"
