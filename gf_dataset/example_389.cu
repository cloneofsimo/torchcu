#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <>

#define CHECK(condition)                                 \
    {                                                    \
        if (!(condition)) {                             \
            fprintf(stderr, "Error: " #condition "\n"); \
            exit(1);                                   \
        }                                                \
    }

extern "C" {

void torch_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input1_data = va_arg(args, const float*);
    int input1_dim0 = va_arg(args, int);
    int input1_dim1 = va_arg(args, int);

    const float* input2_data = va_arg(args, const float*);
    int input2_dim0 = va_arg(args, int);
    int input2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output_data = va_arg(args, float*);

    va_end(args);

    // Allocate device memory
    float *d_input1, *d_input2, *d_output;
    CHECK(hipMalloc(&d_input1, input1_dim0 * input1_dim1 * sizeof(float)));
    CHECK(hipMalloc(&d_input2, input2_dim0 * input2_dim1 * sizeof(float)));
    CHECK(hipMalloc(&d_output, input1_dim0 * input2_dim0 * sizeof(float)));

    // Copy input data to device
    CHECK(hipMemcpy(d_input1, input1_data, input1_dim0 * input1_dim1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_input2, input2_data, input2_dim0 * input2_dim1 * sizeof(float), hipMemcpyHostToDevice));

    // Use cuDNN for pairwise Manhattan distance calculation
    // Create cuDNN context
    hipdnnHandle_t cudnnHandle;
    CHECK(hipdnnCreate(&cudnnHandle));

    // Create cuDNN tensor descriptors
    hipdnnTensorDescriptor_t input1Desc, input2Desc, outputDesc;
    CHECK(hipdnnCreateTensorDescriptor(&input1Desc));
    CHECK(hipdnnCreateTensorDescriptor(&input2Desc));
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc));

    // Set tensor descriptors
    CHECK(hipdnnSetTensorNdDescriptor(input1Desc, HIPDNN_DATA_FLOAT, 1, &input1_dim0));
    CHECK(hipdnnSetTensorNdDescriptor(input2Desc, HIPDNN_DATA_FLOAT, 1, &input2_dim0));
    CHECK(hipdnnSetTensorNdDescriptor(outputDesc, HIPDNN_DATA_FLOAT, 2, &input1_dim0, &input2_dim0));

    // Create cuDNN operation descriptor
    hipdnnOpTensorDescriptor_t opDesc;
    CHECK(hipdnnCreateOpTensorDescriptor(&opDesc));
    CHECK(hipdnnSetOpTensorDescriptor(opDesc, HIPDNN_OP_TENSOR_ADD,
                                     HIPDNN_OP_TENSOR_MUL, HIPDNN_PROPAGATE_NAN));

    // Perform pairwise Manhattan distance calculation using cuDNN
    // We use abs(input1 - input2).sum(dim=-1)
    // This can be achieved by:
    // 1. input1 - input2 (broadcasting)
    // 2. |input1 - input2|
    // 3. sum(dim=-1)
    // We can achieve 1 and 2 by using HIPDNN_OP_TENSOR_ADD with
    // input1, -input2 and output as the first, second, and third
    // arguments. 
    // For 3, we can simply call cudnnTransformTensorEx directly on the
    // output. 

    // 1. Broadcast subtraction (input1 - input2)
    CHECK(cudnnTransformTensorEx(cudnnHandle, opDesc, input1Desc, d_input1,
                                   input2Desc, d_input2, 
                                   outputDesc, d_output));

    // 2. Take absolute value (|input1 - input2|)
    CHECK(cudnnTransformTensorEx(cudnnHandle, opDesc, outputDesc, d_output,
                                   outputDesc, d_output, 
                                   outputDesc, d_output));

    // 3. Sum along the last dimension (sum(dim=-1))
    CHECK(cudnnTransformTensorEx(cudnnHandle, opDesc, outputDesc, d_output,
                                   outputDesc, d_output, 
                                   outputDesc, d_output));

    // Copy result back to host
    CHECK(hipMemcpy(output_data, d_output, input1_dim0 * input2_dim0 * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK(hipFree(d_input1));
    CHECK(hipFree(d_input2));
    CHECK(hipFree(d_output));

    // Free cuDNN resources
    CHECK(hipdnnDestroyTensorDescriptor(input1Desc));
    CHECK(hipdnnDestroyTensorDescriptor(input2Desc));
    CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CHECK(hipdnnDestroyOpTensorDescriptor(opDesc));
    CHECK(hipdnnDestroy(cudnnHandle));
}

}  // extern "C"
