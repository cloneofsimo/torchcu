
#include <hip/hip_runtime.h>
#include <hipDNN.h>

extern "C" {

void torch_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    const float* vec = va_arg(args, const float*);
    int vec_dim = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Check if input dimensions are compatible
    if (input_tensor_dim1 != weight_dim1 || weight_dim0 != vec_dim) {
        // Handle error: incompatible dimensions
        return;
    }

    // Cudnn setup
    hipdnnHandle_t cudnn_handle;
    hipdnnCreate(&cudnn_handle);

    // Allocate device memory
    float *d_input, *d_weight, *d_vec, *d_output;
    hipMalloc(&d_input, input_tensor_dim0 * input_tensor_dim1 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * weight_dim1 * sizeof(float));
    hipMalloc(&d_vec, vec_dim * sizeof(float));
    hipMalloc(&d_output, input_tensor_dim0 * input_tensor_dim1 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * input_tensor_dim1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec, vec_dim * sizeof(float), hipMemcpyHostToDevice);

    // Create cudnn tensor descriptors
    hipdnnTensorDescriptor_t input_tensor_desc, weight_desc, vec_desc, output_desc;
    hipdnnCreateTensorDescriptor(&input_tensor_desc);
    hipdnnCreateTensorDescriptor(&weight_desc);
    hipdnnCreateTensorDescriptor(&vec_desc);
    hipdnnCreateTensorDescriptor(&output_desc);

    // Set tensor descriptors
    hipdnnSetTensorNdDescriptor(input_tensor_desc, HIPDNN_DATA_FLOAT, 2,
                                 (const int[]){input_tensor_dim0, input_tensor_dim1});
    hipdnnSetTensorNdDescriptor(weight_desc, HIPDNN_DATA_FLOAT, 2,
                                 (const int[]){weight_dim0, weight_dim1});
    hipdnnSetTensorNdDescriptor(vec_desc, HIPDNN_DATA_FLOAT, 1,
                                 (const int[]){vec_dim});
    hipdnnSetTensorNdDescriptor(output_desc, HIPDNN_DATA_FLOAT, 2,
                                 (const int[]){input_tensor_dim0, input_tensor_dim1});

    // Perform addmv operation
    hipdnnAddTensor(cudnn_handle, CUDNN_ADD_SAME_ALPHA_BETA,
                      &one, input_tensor_desc, d_input,
                      &one, weight_desc, d_weight, 
                      &one, vec_desc, d_vec, 
                      &one, output_desc, d_output);

    // Copy result back to host
    hipMemcpy(output, d_output, input_tensor_dim0 * input_tensor_dim1 * sizeof(float), hipMemcpyDeviceToHost);

    // Free resources
    hipdnnDestroyTensorDescriptor(input_tensor_desc);
    hipdnnDestroyTensorDescriptor(weight_desc);
    hipdnnDestroyTensorDescriptor(vec_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_vec);
    hipFree(d_output);
    hipdnnDestroy(cudnn_handle);
}

}  // extern "C"
