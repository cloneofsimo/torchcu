#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// CUDA kernel for Hadamard product and sigmoid
__global__ void hadamard_sigmoid_kernel(const float* input_tensor, const float* weight, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = __expf(-__fmaf(-input_tensor[idx] * weight[idx], 1.0f, 0.0f)) / (1.0f + __expf(-__fmaf(-input_tensor[idx] * weight[idx], 1.0f, 0.0f)));
    }
}

extern "C" {

void torch_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * input_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int size = batch_size * input_dim;
    dim3 threadsPerBlock(1024);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hadamard_sigmoid_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, size
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * input_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
