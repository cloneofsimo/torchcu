#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/epilogue/threadblock/epilogue.h>
#include <cutlass/layout/tensor.h>
#include <cutlass/util/tensor_view.h>
#include <cutlass/transform/threadblock/transform.h>
#include <cutlass/epilogue/threadblock/fast_int8.h>
#include <cutlass/epilogue/threadblock/fast_fp16.h>
#include <cutlass/epilogue/threadblock/linear_combination.h>

#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

#define kThreadblockRows 128
#define kThreadblockCols 16
#define kWarpSize 32
#define kSMEM_BYTES (kThreadblockRows * kThreadblockCols * sizeof(int8_t))

using namespace cutlass;

// CUDA kernel for matrix multiplication and ReLU using bfloat16
__global__ void token_mixing_function_kernel(
    const int8_t* input_tensor,
    const int8_t* weight_qkv,
    const int8_t* weight_out,
    const int8_t* norm_weight,
    const int8_t* norm_bias,
    float* output,
    int batch_size,
    int seq_len,
    int d_model
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batch_size && col < seq_len) {
        int8_t* smem = reinterpret_cast<int8_t*>(__ldg(&shared_mem[0]));

        // LayerNorm
        int8_t* norm_input = reinterpret_cast<int8_t*>(input_tensor + (row * seq_len + col) * d_model);
        float norm_sum = 0.0f;
        for (int i = 0; i < d_model; i++) {
            norm_sum += norm_input[i] * norm_weight[i];
        }
        float norm_result = (norm_sum + norm_bias[0]) / sqrtf(d_model);

        // Multiply by qkv weight
        for (int i = 0; i < d_model; i++) {
            smem[threadIdx.y * d_model + i] = norm_input[i] * weight_qkv[col * d_model * 3 + i];
        }
        __syncthreads();

        // Attention
        float q[kThreadblockCols] = {0};
        float k[kThreadblockCols] = {0};
        float v[kThreadblockCols] = {0};

        for (int i = 0; i < kThreadblockCols; i++) {
            q[i] = smem[threadIdx.y * d_model + i];
            k[i] = smem[threadIdx.y * d_model + i + d_model];
            v[i] = smem[threadIdx.y * d_model + i + 2 * d_model];
        }

        float attn[kThreadblockCols] = {0};
        for (int i = 0; i < kThreadblockCols; i++) {
            for (int j = 0; j < kThreadblockCols; j++) {
                attn[i] += q[i] * k[j] / sqrtf(d_model);
            }
        }

        // Softmax
        float sum = 0.0f;
        for (int i = 0; i < kThreadblockCols; i++) {
            sum += expf(attn[i]);
        }
        for (int i = 0; i < kThreadblockCols; i++) {
            attn[i] = expf(attn[i]) / sum;
        }

        // Apply attention
        float result[kThreadblockCols] = {0};
        for (int i = 0; i < kThreadblockCols; i++) {
            for (int j = 0; j < kThreadblockCols; j++) {
                result[i] += attn[i] * v[j];
            }
        }

        // Multiply by weight_out
        float output_sum = 0.0f;
        for (int i = 0; i < kThreadblockCols; i++) {
            output_sum += result[i] * weight_out[i * d_model + threadIdx.x];
        }
        output[(row * seq_len + col) * d_model + threadIdx.x] = output_sum;
    }
}

extern "C" {
void token_mixing_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const int8_t* input_tensor = va_arg(args, const int8_t*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);

    // Extract weight_qkv tensor
    const int8_t* weight_qkv = va_arg(args, const int8_t*);
    int weight_qkv_dim0 = va_arg(args, int);
    int weight_qkv_dim1 = va_arg(args, int);

    // Extract weight_out tensor
    const int8_t* weight_out = va_arg(args, const int8_t*);
    int weight_out_dim0 = va_arg(args, int);
    int weight_out_dim1 = va_arg(args, int);

    // Extract norm_weight tensor
    const int8_t* norm_weight = va_arg(args, const int8_t*);
    int norm_weight_dim0 = va_arg(args, int);

    // Extract norm_bias tensor
    const int8_t* norm_bias = va_arg(args, const int8_t*);
    int norm_bias_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Calculate kernel dimensions
    int batch_size = input_tensor_dim0;
    int seq_len = input_tensor_dim1;
    int d_model = input_tensor_dim2;

    // Allocate device memory
    int8_t *d_input, *d_weight_qkv, *d_weight_out, *d_norm_weight, *d_norm_bias;
    float *d_output;
    hipMalloc(&d_input, batch_size * seq_len * d_model * sizeof(int8_t));
    hipMalloc(&d_weight_qkv, weight_qkv_dim0 * weight_qkv_dim1 * sizeof(int8_t));
    hipMalloc(&d_weight_out, weight_out_dim0 * weight_out_dim1 * sizeof(int8_t));
    hipMalloc(&d_norm_weight, norm_weight_dim0 * sizeof(int8_t));
    hipMalloc(&d_norm_bias, norm_bias_dim0 * sizeof(int8_t));
    hipMalloc(&d_output, batch_size * seq_len * d_model * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * seq_len * d_model * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_weight_qkv, weight_qkv, weight_qkv_dim0 * weight_qkv_dim1 * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_weight_out, weight_out, weight_out_dim0 * weight_out_dim1 * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_norm_weight, norm_weight, norm_weight_dim0 * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_norm_bias, norm_bias, norm_bias_dim0 * sizeof(int8_t), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(kThreadblockCols, kThreadblockRows);
    dim3 numBlocks((seq_len + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    token_mixing_function_kernel<<<numBlocks, threadsPerBlock, kSMEM_BYTES>>>(
        d_input, d_weight_qkv, d_weight_out, d_norm_weight, d_norm_bias, d_output,
        batch_size, seq_len, d_model
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * seq_len * d_model * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight_qkv);
    hipFree(d_weight_out);
    hipFree(d_norm_weight);
    hipFree(d_norm_bias);
    hipFree(d_output);
}

} // extern "C"
