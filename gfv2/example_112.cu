
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for the soft shrink and cross-fade operation
__global__ void softshrink_crossfade_kernel(const float* input_tensor, float threshold, const float* weight, float alpha, float* output, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float value = input_tensor[i];
        if (abs(value) > threshold) {
            output[i] = (value - threshold * (value > 0.0f ? 1.0f : -1.0f)) * (1 - alpha) + weight[i] * alpha;
        } else {
            output[i] = weight[i] * alpha;
        }
    }
}

extern "C" {

void softshrink_crossfade_fp32(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract threshold
    float threshold = va_arg(args, float);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract alpha
    float alpha = va_arg(args, float);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input_tensor_dim0 * input_tensor_dim1;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_weight, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    softshrink_crossfade_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, threshold, d_weight, alpha, d_output, size
    );

    // Copy result back to host
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
