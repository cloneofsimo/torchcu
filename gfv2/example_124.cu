
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include <stdarg.h>

extern "C" {

void inverse_fourier_transform_fp32(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    float* output = va_arg(args, float*);

    va_end(args);

    // Input dimensions
    int batch_size = input_tensor_dim0;
    int channels = input_tensor_dim1;
    int dim2 = input_tensor_dim2;
    int dim3 = input_tensor_dim3;

    // Calculate the size of each complex element (real + imaginary)
    int complex_element_size = sizeof(float) * 2;

    // Calculate total input size in bytes
    size_t input_size_bytes = batch_size * channels * dim2 * dim3 * complex_element_size;

    // Allocate device memory for input
    float *d_input;
    hipMalloc(&d_input, input_size_bytes);

    // Allocate device memory for output
    float *d_output;
    hipMalloc(&d_output, batch_size * channels * dim2 * dim3 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_size_bytes, hipMemcpyHostToDevice);

    // Set up cuFFT plan
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, &dim3, &dim2, 1, &channels,
                  &dim3, &dim2, 1, &channels, HIPFFT_C2R, batch_size);

    // Execute the inverse transform
    hipfftExecC2R(plan, (hipfftComplex*)d_input, d_output);

    // Destroy the plan
    hipfftDestroy(plan);

    // Copy output data back to host
    hipMemcpy(output, d_output, batch_size * channels * dim2 * dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
