#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

// CUDA kernel for optimized ReLU with scaling
__global__ void optimized_relu_kernel(const float* input, float* output, float alpha, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = fmaxf(input[idx], 0.0f) * alpha;  // ReLU and scaling in one step
    }
}

extern "C" {

void hyperparameter_optimized_relu(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    // Extract alpha
    float alpha = va_arg(args, float);

    // Extract output tensor
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input_dim0 * input_dim1;

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    optimized_relu_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, alpha, size);

    // Copy result back to host
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
