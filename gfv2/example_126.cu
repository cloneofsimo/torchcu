#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <math.h>

// Helper function to compute softmax along a dimension
__device__ __forceinline__ float softmax(float* x, int size) {
    float max_val = x[0];
    for (int i = 1; i < size; ++i) {
        max_val = fmaxf(max_val, x[i]);
    }

    float sum = 0.0f;
    for (int i = 0; i < size; ++i) {
        x[i] = expf(x[i] - max_val);
        sum += x[i];
    }

    for (int i = 0; i < size; ++i) {
        x[i] /= sum;
    }

    return max_val;
}

// CUDA kernel for local attention
__global__ void local_attention_kernel(float* input, float* output, int B, int T, int C, int window_size, int causal) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && t < T) {
        // Calculate start and end indices for local window
        int start_idx = t - window_size + 1;
        int end_idx = t + window_size;

        // Apply causal masking
        if (causal) {
            end_idx = min(end_idx, t + window_size);
        }

        // Clamp indices to valid range
        start_idx = max(start_idx, 0);
        end_idx = min(end_idx, T);

        int window_size_eff = end_idx - start_idx;

        // Calculate offsets for local window
        int input_offset = b * T * C + t * C;
        int output_offset = b * T * C + t * C;

        // Compute Q, K, V for local window
        float* q = input + input_offset;
        float* k = input + (b * T * C + start_idx * C);
        float* v = input + (b * T * C + start_idx * C);

        // Calculate attention weights
        float attn[window_size_eff * C];
        for (int i = 0; i < window_size_eff; ++i) {
            for (int j = 0; j < C; ++j) {
                attn[i * C + j] = q[j] * k[i * C + j];
            }
        }
        for (int i = 0; i < window_size_eff * C; ++i) {
            attn[i] /= sqrtf(C);
        }

        // Apply causal masking if required
        if (causal) {
            for (int i = 0; i < window_size_eff; ++i) {
                for (int j = 0; j < i; ++j) {
                    attn[i * C + j] = -INFINITY;
                }
            }
        }

        // Compute softmax
        float max_val = softmax(attn, window_size_eff * C);

        // Apply attention
        for (int i = 0; i < C; ++i) {
            float sum = 0.0f;
            for (int j = 0; j < window_size_eff; ++j) {
                sum += attn[j * C + i] * v[j * C + i];
            }
            output[output_offset + i] = sum;
        }
    }
}

extern "C" {

void local_attention_forward(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input = va_arg(args, const float*);
    int B = va_arg(args, int);
    int T = va_arg(args, int);
    int C = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    // Extract window size and causal flag
    int window_size = va_arg(args, int);
    int causal = va_arg(args, int);

    va_end(args);

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, B * T * C * sizeof(float));
    hipMalloc(&d_output, B * T * C * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input, B * T * C * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((B + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (T + threadsPerBlock.y - 1) / threadsPerBlock.y);

    local_attention_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_output, B, T, C, window_size, causal
    );

    // Copy result back to host
    hipMemcpy(output, d_output, B * T * C * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
