#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <stdarg.h>

__global__ void harmonic_percussive_separation_int8_kernel(const int8_t* audio, int8_t* harmonic, int8_t* percussive, int n, int t, int iterations) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < t) {
        int8_t* h = harmonic + i;
        int8_t* p = percussive + i;
        int8_t* a = audio + i;
        *h = 0;
        *p = 0;

        for (int iter = 0; iter < iterations; ++iter) {
            int8_t cross_correlation = 0;
            for (int j = 0; j < t; ++j) {
                cross_correlation += *h * *(p + (t - j - 1));
            }

            *h = (int8_t) ((float)(*h + *a + cross_correlation) * 0.5f);
            *p = (int8_t) ((float)(*p + *a - cross_correlation) * 0.5f);
        }
    }
}

extern "C" {

void harmonic_percussive_separation_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* audio = va_arg(args, const float*);
    int n = va_arg(args, int);
    int t = va_arg(args, int);
    int iterations = va_arg(args, int);

    float* harmonic = va_arg(args, float*);

    va_end(args);

    int8_t* d_audio;
    int8_t* d_harmonic;
    int8_t* d_percussive;

    hipMalloc(&d_audio, t * sizeof(int8_t));
    hipMalloc(&d_harmonic, t * sizeof(int8_t));
    hipMalloc(&d_percussive, t * sizeof(int8_t));

    hipMemcpy(d_audio, audio, t * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(128);
    dim3 numBlocks((t + threadsPerBlock.x - 1) / threadsPerBlock.x);

    harmonic_percussive_separation_int8_kernel<<<numBlocks, threadsPerBlock>>>(d_audio, d_harmonic, d_percussive, n, t, iterations);

    hipMemcpy(harmonic, d_harmonic, t * sizeof(int8_t), hipMemcpyDeviceToHost);

    hipFree(d_audio);
    hipFree(d_harmonic);
    hipFree(d_percussive);
}

}
