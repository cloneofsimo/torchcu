#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <math.h>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// Swiglu activation function
__device__ __forceinline__ float swiglu(float x) {
    return x * 1.0f / (1.0f + expf(-x));
}

// Kernel for matrix multiplication and Swiglu activation
__global__ void matmul_swiglu_kernel_bf16(const float* input_tensor, const float* weight, float* output, 
                                            int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            __hip_bfloat16 a = float_to_bfloat16(input_tensor[row * k + i]);
            __hip_bfloat16 b = float_to_bfloat16(weight[col * k + i]);
            sum += bfloat16_to_float(__hmul(a, b));
        }
        output[row * n + col] = swiglu(sum);
    }
}

// Kernel for channel attention
__global__ void channel_attention_kernel(const float* output, const float* channel_attention_weight, float* result,
                                        int batch_size, int output_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batch_size * output_dim) {
        int batch = idx / output_dim;
        int feature = idx % output_dim;
        result[idx] = output[idx] * channel_attention_weight[batch];
    }
}

// Kernel for calculating the ArcFace loss
__global__ void arcface_loss_kernel(const float* output, const int* target, float* loss, int batch_size, 
                                      int output_dim, float s, float m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batch_size) {
        int label = target[idx];
        float max_score = -FLT_MAX;
        for (int j = 0; j < output_dim; ++j) {
            if (j == label) {
                max_score = output[idx * output_dim + j];
            }
        }
        
        float phi = max_score * s;
        phi = phi + m;
        
        loss[idx] = -logf(expf(phi) / (expf(phi) + expf(output[idx * output_dim + label])));
    }
}

extern "C" {

void forward_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight_tensor = va_arg(args, const float*);
    int weight_tensor_dim0 = va_arg(args, int);
    int weight_tensor_dim1 = va_arg(args, int);

    // Extract target tensor
    const int* target = va_arg(args, const int*);
    int target_dim = va_arg(args, int);

    // Extract channel attention weight tensor
    const float* channel_attention_weight = va_arg(args, const float*);
    int channel_attention_weight_dim0 = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);
    
    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_tensor_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_output, *d_channel_attention_weight;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));
    hipMalloc(&d_channel_attention_weight, channel_attention_weight_dim0 * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight_tensor, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_channel_attention_weight, channel_attention_weight, channel_attention_weight_dim0 * sizeof(float), hipMemcpyHostToDevice);

    // Launch matrix multiplication and Swiglu kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matmul_swiglu_kernel_bf16<<<numBlocks, threadsPerBlock>>>(d_input, d_weight, d_output, batch_size, output_dim, input_dim);

    // Launch channel attention kernel
    numBlocks = (batch_size * output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x;
    channel_attention_kernel<<<numBlocks, threadsPerBlock>>>(d_output, d_channel_attention_weight, d_output, batch_size, output_dim);

    // Calculate ArcFace loss
    float s = 64.0f;
    float m = 0.50f;
    float *d_loss;
    hipMalloc(&d_loss, batch_size * sizeof(float));
    arcface_loss_kernel<<<(batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x, threadsPerBlock>>>(d_output, target, d_loss, batch_size, output_dim, s, m);

    // Copy loss to host
    hipMemcpy(output, d_loss, batch_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    hipFree(d_channel_attention_weight);
    hipFree(d_loss);
}

}  // extern "C"
