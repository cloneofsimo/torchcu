#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include <stdarg.h>

extern "C" {

__global__ void diagflat_multiply_kernel(const float* input, const float* weight, float* output,
                                         int batch_size, int channels, int width, int diag_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size * channels * width) {
        int b = idx / (channels * width);
        int c = (idx % (channels * width)) / width;
        int w = idx % width;

        float sum = 0.0f;
        for (int i = 0; i < diag_size; ++i) {
            sum += input[(b * channels + c) * width + (w + i) % width] * weight[i];
        }
        output[idx] = sum;
    }
}

void conv_fft_diagflat_inplace(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);
    int input_dim2 = va_arg(args, int);
    int input_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);

    va_end(args);

    // Allocate device memory
    float* d_input, *d_weight, *d_output;
    hipMalloc(&d_input, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * sizeof(float));
    hipMalloc(&d_output, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * sizeof(float), hipMemcpyHostToDevice);

    // Perform convolution using FFT on device
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, &input_dim3, &input_dim3, 1, input_dim3 * sizeof(float), 0, 0,
                  1, &input_dim3, &input_dim3, 1, input_dim3 * sizeof(float), 0, 0,
                  HIPFFT_C2C, input_dim0 * input_dim1 * input_dim2);

    // Allocate complex data on device
    hipfftComplex* d_input_complex;
    hipMalloc(&d_input_complex, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(hipfftComplex));

    // Copy real data to complex data (assuming real input)
    hipMemcpy(d_input_complex, d_input, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(float), hipMemcpyDeviceToDevice);

    // Perform forward FFT
    hipfftExecC2C(plan, (hipfftComplex*)d_input_complex, (hipfftComplex*)d_input_complex, HIPFFT_FORWARD);

    // Perform diagonal matrix multiplication
    diagflat_multiply_kernel<<<(input_dim0 * input_dim1 * input_dim2 * input_dim3 + 128 - 1) / 128, 128>>>(
        d_input_complex, d_weight, d_output, input_dim0, input_dim1, input_dim3, weight_dim0);

    // Perform inverse FFT
    hipfftExecC2C(plan, (hipfftComplex*)d_output, (hipfftComplex*)d_output, HIPFFT_BACKWARD);

    // Copy result back to host
    hipMemcpy(d_input, d_output, input_dim0 * input_dim1 * input_dim2 * input_dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipfftDestroy(plan);
    hipFree(d_input_complex);
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

} // extern "C"
