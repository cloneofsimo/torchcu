#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for SEBlock
__global__ void se_block_kernel(const float* input, float* output, int batch_size, int channels, 
                                int height, int width, int reduction) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < batch_size && c < channels) {
        float sum = 0.0f;
        for (int h = 0; h < height; ++h) {
            for (int w = 0; w < width; ++w) {
                __hip_bfloat16 val = float_to_bfloat16(input[(b * channels + c) * height * width + h * width + w]);
                sum += bfloat16_to_float(val);
            }
        }
        sum /= height * width;

        __hip_bfloat16 reduced = float_to_bfloat16(sum);
        reduced = __hmul(reduced, reduced);  // Square
        reduced = __hmul(reduced, reduced);  // Square again
        reduced = __hmul(reduced, reduced);  // Square again
        reduced = __hmul(reduced, reduced);  // Square again

        // Apply sigmoid
        reduced = __hmul(reduced, __float2bfloat16(1.0f) + reduced);  // 1 + x
        reduced = __hmul(reduced, __float2bfloat16(0.5f));  // (1 + x) / 2

        for (int h = 0; h < height; ++h) {
            for (int w = 0; w < width; ++w) {
                __hip_bfloat16 val = float_to_bfloat16(input[(b * channels + c) * height * width + h * width + w]);
                output[(b * channels + c) * height * width + h * width + w] = 
                    bfloat16_to_float(__hmul(val, reduced));
            }
        }
    }
}

// CUDA kernel for subtraction
__global__ void subtract_kernel(const float* se_output, const float* weight, float* output, 
                                int batch_size, int channels, int height, int width) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;
    int h = blockIdx.z * blockDim.z + threadIdx.z;
    int w = threadIdx.w;

    if (b < batch_size && c < channels && h < height && w < width) {
        output[(b * channels + c) * height * width + h * width + w] =
            se_output[(b * channels + c) * height * width + h * width + w] - weight[c];
    }
}

extern "C" {

void se_subtract_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int channels = input_tensor_dim1;
    int height = input_tensor_dim2;
    int width = input_tensor_dim3;
    int reduction = channels / 16; // Default reduction for SEBlock

    // Allocate device memory
    float *d_input, *d_se_output, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * channels * height * width * sizeof(float));
    hipMalloc(&d_se_output, batch_size * channels * height * width * sizeof(float));
    hipMalloc(&d_weight, channels * sizeof(float));
    hipMalloc(&d_output, batch_size * channels * height * width * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * channels * height * width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, channels * sizeof(float), hipMemcpyHostToDevice);

    // Launch SEBlock kernel
    dim3 se_block_threads(16, 16);
    dim3 se_block_blocks((channels + se_block_threads.y - 1) / se_block_threads.y, 
                           (batch_size + se_block_threads.x - 1) / se_block_threads.x);
    se_block_kernel<<<se_block_blocks, se_block_threads>>>(d_input, d_se_output, batch_size, channels, 
                                                         height, width, reduction);

    // Launch subtraction kernel
    dim3 subtract_threads(16, 16, 4);
    dim3 subtract_blocks((batch_size + subtract_threads.x - 1) / subtract_threads.x, 
                          (channels + subtract_threads.y - 1) / subtract_threads.y, 
                          (height + subtract_threads.z - 1) / subtract_threads.z);
    subtract_kernel<<<subtract_blocks, subtract_threads>>>(d_se_output, d_weight, d_output, 
                                                           batch_size, channels, height, width);

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * channels * height * width * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_se_output);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
