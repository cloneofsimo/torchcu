#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void logsumexp_loss_kernel(const float* input, const float* target, float* output, int batch_size, int num_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        float lse = -INFINITY;
        for (int i = 0; i < num_classes; ++i) {
            lse = fmaxf(lse, input[idx * num_classes + i]);
        }
        float loss = lse;
        for (int i = 0; i < num_classes; ++i) {
            loss -= input[idx * num_classes + i] * target[idx * num_classes + i];
        }
        output[idx] = loss;
    }
}

extern "C" {

void logsumexp_loss_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* input = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    const float* target = va_arg(args, const float*);
    int target_dim0 = va_arg(args, int);
    int target_dim1 = va_arg(args, int);

    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_dim0;
    int num_classes = input_dim1;

    float *d_input, *d_target, *d_output;
    hipMalloc(&d_input, batch_size * num_classes * sizeof(float));
    hipMalloc(&d_target, batch_size * num_classes * sizeof(float));
    hipMalloc(&d_output, batch_size * sizeof(float));

    hipMemcpy(d_input, input, batch_size * num_classes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, batch_size * num_classes * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    logsumexp_loss_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_target, d_output, batch_size, num_classes
    );

    hipMemcpy(output, d_output, batch_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_target);
    hipFree(d_output);
}

}  // extern "C"
