#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <math.h>
#include <stdarg.h>

// Helper function to convert float to half
__device__ __forceinline__ half float_to_half(float f) {
    return __float2half_rn(f);
}

// Helper function to convert half to float
__device__ __forceinline__ float half_to_float(half h) {
    return __half2float(h);
}

__global__ void normalize_kernel(const float* input, half* output, int batch_size, int embedding_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size * embedding_dim) {
        int batch_idx = idx / embedding_dim;
        int feature_idx = idx % embedding_dim;
        float sum_sq = 0.0f;
        for (int i = 0; i < embedding_dim; ++i) {
            sum_sq += input[batch_idx * embedding_dim + i] * input[batch_idx * embedding_dim + i];
        }
        output[idx] = float_to_half(input[idx] / sqrtf(sum_sq));
    }
}

__global__ void simclr_loss_kernel(const half* z1, const half* z2, float* loss, int batch_size, int embedding_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        float max_similarity = -1e9f;
        float positive_similarity = -1e9f;
        for (int i = 0; i < batch_size; ++i) {
            float similarity = 0.0f;
            for (int j = 0; j < embedding_dim; ++j) {
                similarity += half_to_float(z1[idx * embedding_dim + j]) * half_to_float(z2[i * embedding_dim + j]);
            }
            if (i == idx) {
                positive_similarity = similarity;
            } else if (similarity > max_similarity) {
                max_similarity = similarity;
            }
        }
        loss[idx] = -logf(expf(positive_similarity) / (expf(positive_similarity) + expf(max_similarity)));
    }
}

extern "C" {

void simclr_loss_fp16(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* z1 = va_arg(args, const float*);
    int z1_dim0 = va_arg(args, int);
    int z1_dim1 = va_arg(args, int);

    const float* z2 = va_arg(args, const float*);
    int z2_dim0 = va_arg(args, int);
    int z2_dim1 = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = z1_dim0;
    int embedding_dim = z1_dim1;

    // Allocate device memory
    half* d_z1, *d_z2;
    float* d_loss;
    hipMalloc(&d_z1, batch_size * embedding_dim * sizeof(half));
    hipMalloc(&d_z2, batch_size * embedding_dim * sizeof(half));
    hipMalloc(&d_loss, batch_size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_z1, z1, batch_size * embedding_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z2, z2, batch_size * embedding_dim * sizeof(float), hipMemcpyHostToDevice);

    // Normalize embeddings on the device
    dim3 threadsPerBlock(256);
    dim3 numBlocks((batch_size * embedding_dim + threadsPerBlock.x - 1) / threadsPerBlock.x);
    normalize_kernel<<<numBlocks, threadsPerBlock>>>(d_z1, d_z1, batch_size, embedding_dim);
    normalize_kernel<<<numBlocks, threadsPerBlock>>>(d_z2, d_z2, batch_size, embedding_dim);

    // Calculate SimCLR loss on the device
    numBlocks = (batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    simclr_loss_kernel<<<numBlocks, threadsPerBlock>>>(d_z1, d_z2, d_loss, batch_size, embedding_dim);

    // Copy result back to host
    hipMemcpy(output, d_loss, batch_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_z1);
    hipFree(d_z2);
    hipFree(d_loss);
}

}  // extern "C"
