#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

__global__ void zero_crossing_rate_kernel(const int8_t* input_tensor, float* output,
                                        int batch_size, int channels, int length) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < batch_size && c < channels) {
        float zcr = 0.0f;
        for (int i = 1; i < length; ++i) {
            if ((input_tensor[b * channels * length + c * length + i] * 
                 input_tensor[b * channels * length + c * length + i - 1]) < 0) {
                zcr += 1.0f;
            }
        }
        output[b * channels + c] = zcr / (length - 1);
    }
}

extern "C" {

void zero_crossing_rate_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor_float = va_arg(args, const float*);
    int batch_size = va_arg(args, int);
    int channels = va_arg(args, int);
    int length = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Allocate device memory for input and output
    int8_t* d_input;
    hipMalloc(&d_input, batch_size * channels * length * sizeof(int8_t));

    // Copy input data to device as int8
    hipMemcpy(d_input, input_tensor_float, batch_size * channels * length * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (channels + threadsPerBlock.y - 1) / threadsPerBlock.y);

    zero_crossing_rate_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, output, batch_size, channels, length
    );

    // Free device memory
    hipFree(d_input);
}

}  // extern "C"
