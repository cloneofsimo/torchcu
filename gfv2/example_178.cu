#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <math.h>  // For expf
#include <stdarg.h>

// Helper functions for bfloat16 conversion
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// Helper function for calculating IoU
__device__ float calculate_iou(float* box1, float* box2) {
    float x1 = max(box1[0], box2[0]);
    float y1 = max(box1[1], box2[1]);
    float x2 = min(box1[2], box2[2]);
    float y2 = min(box1[3], box2[3]);

    float intersection_area = max(0.0f, x2 - x1) * max(0.0f, y2 - y1);
    float box1_area = (box1[2] - box1[0]) * (box1[3] - box1[1]);
    float box2_area = (box2[2] - box2[0]) * (box2[3] - box2[1]);
    float union_area = box1_area + box2_area - intersection_area;

    return intersection_area / union_area;
}

// CUDA kernel for NMS with exponential scoring
__global__ void nms_kernel_bf16(const float* boxes, const float* scores, float* keep,
                                 int num_boxes, float iou_threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_boxes) {
        if (keep[idx] == 1.0f) {  // Check if box is already suppressed
            for (int j = idx + 1; j < num_boxes; ++j) {
                if (keep[j] == 1.0f) {
                    __hip_bfloat16 score_i = float_to_bfloat16(scores[idx]);
                    __hip_bfloat16 score_j = float_to_bfloat16(scores[j]);
                    __hip_bfloat16 iou = float_to_bfloat16(calculate_iou(boxes + idx * 4, boxes + j * 4));

                    if (iou > iou_threshold && score_j > score_i) {
                        keep[idx] = 0.0f;
                        break;  // Exit the inner loop if the box is suppressed
                    }
                }
            }
        }
    }
}

extern "C" {

void nms_exponential_bf16(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* boxes = va_arg(args, const float*);
    int boxes_dim0 = va_arg(args, int);
    int boxes_dim1 = va_arg(args, int);
    const float* scores = va_arg(args, const float*);
    int scores_dim0 = va_arg(args, int);

    // Extract IOU threshold
    float iou_threshold = va_arg(args, float);

    // Extract output tensor (assuming it's preallocated)
    float* keep = va_arg(args, float*);

    va_end(args);

    int num_boxes = boxes_dim0;

    // Allocate device memory
    float *d_boxes, *d_scores, *d_keep;
    hipMalloc(&d_boxes, num_boxes * boxes_dim1 * sizeof(float));
    hipMalloc(&d_scores, num_boxes * sizeof(float));
    hipMalloc(&d_keep, num_boxes * sizeof(float));

    // Copy data to device
    hipMemcpy(d_boxes, boxes, num_boxes * boxes_dim1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_scores, scores, num_boxes * sizeof(float), hipMemcpyHostToDevice);

    // Initialize 'keep' array to 1.0f on the device
    hipMemset(d_keep, 1.0f, num_boxes * sizeof(float));

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (num_boxes + threadsPerBlock - 1) / threadsPerBlock;
    nms_kernel_bf16<<<numBlocks, threadsPerBlock>>>(d_boxes, d_scores, d_keep, num_boxes, iou_threshold);

    // Copy result back to host
    hipMemcpy(keep, d_keep, num_boxes * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_boxes);
    hipFree(d_scores);
    hipFree(d_keep);
}

} // extern "C"
