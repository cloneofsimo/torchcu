#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to half
__device__ __forceinline__ half float_to_half(float f) {
    return __float2half_rn(f);
}

// Helper function to convert half to float
__device__ __forceinline__ float half_to_float(half h) {
    return __half2float(h);
}

// CUDA kernel for self-attention with int8 quantization
__global__ void self_attention_int8_kernel(const int* input_tensor, const int* query_weight, const int* key_weight, 
                                          const int* value_weight, float* output, int batch_size, int seq_len, int hidden_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batch_size && col < seq_len) {
        float sum = 0.0f;
        for (int i = 0; i < hidden_size; ++i) {
            half q = float_to_half((float)input_tensor[row * seq_len * hidden_size + col * hidden_size + i]);
            half k = float_to_half((float)query_weight[i * hidden_size + i]);
            half v = float_to_half((float)key_weight[i * hidden_size + i]);

            // Calculate attention scores
            half attention_score = __expf(__fmaf(q, k, -0.5f * logf((float)hidden_size))); // exp(q*k - 0.5 * log(hidden_size))

            // Apply value weights
            v = float_to_half((float)value_weight[i * hidden_size + i]);
            sum += half_to_float(attention_score * v);
        }
        output[row * seq_len * hidden_size + col * hidden_size] = sum;
    }
}

extern "C" {

void self_attention_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const int* input_tensor = va_arg(args, const int*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);

    // Extract query weight tensor
    const int* query_weight = va_arg(args, const int*);
    int query_weight_dim0 = va_arg(args, int);
    int query_weight_dim1 = va_arg(args, int);

    // Extract key weight tensor
    const int* key_weight = va_arg(args, const int*);
    int key_weight_dim0 = va_arg(args, int);
    int key_weight_dim1 = va_arg(args, int);

    // Extract value weight tensor
    const int* value_weight = va_arg(args, const int*);
    int value_weight_dim0 = va_arg(args, int);
    int value_weight_dim1 = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int seq_len = input_tensor_dim1;
    int hidden_size = input_tensor_dim2;

    // Allocate device memory
    int *d_input, *d_query_weight, *d_key_weight, *d_value_weight;
    float *d_output;
    hipMalloc(&d_input, batch_size * seq_len * hidden_size * sizeof(int));
    hipMalloc(&d_query_weight, query_weight_dim0 * query_weight_dim1 * sizeof(int));
    hipMalloc(&d_key_weight, key_weight_dim0 * key_weight_dim1 * sizeof(int));
    hipMalloc(&d_value_weight, value_weight_dim0 * value_weight_dim1 * sizeof(int));
    hipMalloc(&d_output, batch_size * seq_len * hidden_size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * seq_len * hidden_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_query_weight, query_weight, query_weight_dim0 * query_weight_dim1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_key_weight, key_weight, key_weight_dim0 * key_weight_dim1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_value_weight, value_weight, value_weight_dim0 * value_weight_dim1 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((seq_len + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    self_attention_int8_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_query_weight, d_key_weight, d_value_weight, d_output, batch_size, seq_len, hidden_size
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * seq_len * hidden_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_query_weight);
    hipFree(d_key_weight);
    hipFree(d_value_weight);
    hipFree(d_output);
}

}  // extern "C"
