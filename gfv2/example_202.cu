
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdarg.h>

// Helper function to convert float to half
__device__ __forceinline__ half float_to_half(float f) {
    return __float2half_rn(f);
}

// Helper function to convert half to float
__device__ __forceinline__ float half_to_float(half h) {
    return __half2float(h);
}

// CUDA kernel for fused linear (matrix multiplication + bias) and ReLU using int8
__global__ void fused_linear_relu_kernel_int8(const int8_t* input_tensor, const int8_t* weight, const int8_t* bias, 
                                        float* output, int m, int n, int k, int bias_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        int sum = 0;
        for (int i = 0; i < k; ++i) {
            sum += input_tensor[row * k + i] * weight[col * k + i];
        }
        output[row * n + col] = fmaxf(half_to_float(sum + bias[col]), 0.0f);
    }
}

extern "C" {

void fused_linear_int8_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const int8_t* input_tensor = va_arg(args, const int8_t*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const int8_t* weight = va_arg(args, const int8_t*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract bias tensor
    const int8_t* bias = va_arg(args, const int8_t*);
    int bias_size = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    int8_t *d_input, *d_weight, *d_bias;
    float *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(int8_t));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(int8_t));
    hipMalloc(&d_bias, bias_size * sizeof(int8_t));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(int8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, bias_size * sizeof(int8_t), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    fused_linear_relu_kernel_int8<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_bias, d_output, batch_size, output_dim, input_dim, bias_size
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);
}

}  // extern "C"
