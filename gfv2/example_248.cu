#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

__global__ void round_local_attention_mean_int8_kernel(const int8_t* input, const int8_t* query, const int8_t* key, const int8_t* value, 
                                        int batch_size, int seq_len, int head_size, float* output) {

    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < batch_size && h < head_size) {
        float sum = 0.0f;
        for (int i = 0; i < seq_len; ++i) {
            int input_idx = b * seq_len * head_size + i * head_size + h;
            int query_idx = b * seq_len * head_size + i * head_size + h;
            int key_idx = b * seq_len * head_size + i * head_size + h;
            int value_idx = b * seq_len * head_size + i * head_size + h;

            float attention_score = (float)input[input_idx] * (float)query[query_idx] * (float)key[key_idx] * (float)value[value_idx];
            sum += attention_score;
        }

        // Round and mean pooling
        sum = roundf(sum / seq_len);
        output[b * head_size + h] = sum;
    }
}

extern "C" {

void round_local_attention_mean_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    const float* query_tensor = va_arg(args, const float*);
    int query_tensor_dim0 = va_arg(args, int);
    int query_tensor_dim1 = va_arg(args, int);

    const float* key_tensor = va_arg(args, const float*);
    int key_tensor_dim0 = va_arg(args, int);
    int key_tensor_dim1 = va_arg(args, int);

    const float* value_tensor = va_arg(args, const float*);
    int value_tensor_dim0 = va_arg(args, int);
    int value_tensor_dim1 = va_arg(args, int);

    // Extract window_size
    int window_size = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Dimensions
    int batch_size = input_tensor_dim0;
    int seq_len = input_tensor_dim1;
    int head_size = input_tensor_dim1 / window_size;

    // Allocate device memory
    int8_t *d_input, *d_query, *d_key, *d_value;
    float *d_output;
    hipMalloc(&d_input, batch_size * seq_len * head_size * sizeof(int8_t));
    hipMalloc(&d_query, batch_size * seq_len * head_size * sizeof(int8_t));
    hipMalloc(&d_key, batch_size * seq_len * head_size * sizeof(int8_t));
    hipMalloc(&d_value, batch_size * seq_len * head_size * sizeof(int8_t));
    hipMalloc(&d_output, batch_size * head_size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * seq_len * head_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query_tensor, batch_size * seq_len * head_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key_tensor, batch_size * seq_len * head_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value_tensor, batch_size * seq_len * head_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 8);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x, (head_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    round_local_attention_mean_int8_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_query, d_key, d_value, batch_size, seq_len, head_size, d_output
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * head_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_query);
    hipFree(d_key);
    hipFree(d_value);
    hipFree(d_output);
}

}  // extern "C"
