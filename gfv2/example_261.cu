#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

// CUDA kernel for element-wise minimum
__global__ void elementwise_min_kernel(const float* input1, const float* input2, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = fminf(input1[idx], input2[idx]);
    }
}

extern "C" {

void elementwise_min_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input1 = va_arg(args, const float*);
    int input1_dim0 = va_arg(args, int);
    int input1_dim1 = va_arg(args, int);

    const float* input2 = va_arg(args, const float*);
    int input2_dim0 = va_arg(args, int);
    int input2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Calculate the total size of the input tensors
    int size = input1_dim0 * input1_dim1;

    // Allocate device memory
    float *d_input1, *d_input2, *d_output;
    hipMalloc(&d_input1, size * sizeof(float));
    hipMalloc(&d_input2, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input1, input1, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    elementwise_min_kernel<<<numBlocks, threadsPerBlock>>>(d_input1, d_input2, d_output, size);

    // Copy result back to host
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
}

}  // extern "C"
