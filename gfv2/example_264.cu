#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <>

#define BLOCK_SIZE 256

__global__ void max_pool_scatter_add_fp16_kernel(const half* input, const int* indices, const int* lengths, 
                                                half* output, int batch_size, int seq_len, int max_len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch_size) {
        int start = 0;
        int end = lengths[tid];
        for (int i = start; i < end; i++) {
            int idx = tid * seq_len + i;
            if (input[idx] > output[indices[tid] + i]) {
                output[indices[tid] + i] = input[idx];
            }
        }
    }
}

extern "C" {

void max_pool_scatter_add_fp16(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    // Extract indices tensor
    const int* indices = va_arg(args, const int*);
    int indices_dim0 = va_arg(args, int);

    // Extract lengths tensor
    const int* lengths = va_arg(args, const int*);
    int lengths_dim0 = va_arg(args, int);

    // Extract output tensor (pre-allocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Allocate device memory
    half* d_input;
    int* d_indices;
    int* d_lengths;
    half* d_output;
    hipMalloc(&d_input, input_dim0 * input_dim1 * sizeof(half));
    hipMalloc(&d_indices, indices_dim0 * sizeof(int));
    hipMalloc(&d_lengths, lengths_dim0 * sizeof(int));
    hipMalloc(&d_output, lengths_dim0 * input_dim1 * sizeof(half));

    // Copy input data to device
    hipMemcpy(d_input, input, input_dim0 * input_dim1 * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, indices, indices_dim0 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, lengths, lengths_dim0 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int max_len = lengths_dim0 * input_dim1;
    max_pool_scatter_add_fp16_kernel<<<(input_dim0 + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        d_input, d_indices, d_lengths, d_output, input_dim0, input_dim1, max_len);

    // Copy result back to host
    hipMemcpy(output, d_output, max_len * sizeof(half), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_indices);
    hipFree(d_lengths);
    hipFree(d_output);
}

} // extern "C"
