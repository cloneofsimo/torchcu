
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <math.h>
#include <stdarg.h>

// This version uses shared memory for better performance.

__global__ void low_rank_approx_pow_kernel(const float* input_tensor, float* output_tensor, 
                                             int m, int n, int rank, float exponent) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        __shared__ float U_shared[128]; 
        __shared__ float V_shared[128]; 

        // Load U and V sub-blocks into shared memory
        if (threadIdx.x < rank && threadIdx.y < rank) {
            U_shared[threadIdx.x * blockDim.y + threadIdx.y] = input_tensor[row * n + threadIdx.x];
            V_shared[threadIdx.y * blockDim.x + threadIdx.x] = input_tensor[threadIdx.y * n + col]; 
        }

        __syncthreads();

        // Calculate the sum for the output element
        for (int k = 0; k < rank; k++) {
            sum += U_shared[threadIdx.y * blockDim.y + k] * V_shared[k * blockDim.x + threadIdx.x];
        }

        output_tensor[row * n + col] = powf(sum, exponent); 
    }
}

extern "C" {

void low_rank_approx_pow(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* input_tensor = va_arg(args, const float*);
    int m = va_arg(args, int);
    int n = va_arg(args, int);

    int rank = va_arg(args, int);
    float exponent = va_arg(args, float);

    float* output_tensor = va_arg(args, float*);

    va_end(args);

    // Allocate device memory
    float* d_input, *d_output;
    hipMalloc(&d_input, m * n * sizeof(float));
    hipMalloc(&d_output, m * n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    low_rank_approx_pow_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_output, m, n, rank, exponent
    );

    // Copy result back to host
    hipMemcpy(output_tensor, d_output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
