#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdarg.h>

extern "C" {

__global__ void my_function_kernel(const float* input_tensor, float* output_tensor, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        // Convert to fp16
        __half input_fp16 = __float2half_rn(input_tensor[idx]);

        // Non-zero check
        if (input_fp16 != 0) {
            // Clamp to range
            __half clamped_fp16 = __int2half_rn(__float2int_rn(input_fp16) * (1.0f / 32768.0f) * 32767.0f);
            // Update output
            output_tensor[idx] = __half2float(clamped_fp16);
        }
    }
}

void my_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* input_tensor = va_arg(args, const float*);
    int input_dim0 = va_arg(args, int);
    int input_dim1 = va_arg(args, int);

    float* output_tensor = va_arg(args, float*);

    va_end(args);

    int size = input_dim0 * input_dim1;

    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input_tensor, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    my_function_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, size);

    hipMemcpy(output_tensor, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

}
