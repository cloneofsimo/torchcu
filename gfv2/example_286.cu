
#include <hip/hip_runtime.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// CUDA kernel for diagflat operation
__global__ void diagflat_kernel_int8(const int8_t* input_tensor, float* output, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        output[i * (size + 1)] = (float)input_tensor[i];
    }
}

extern "C" {

void diagflat_int8_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input_tensor_dim0;

    // Allocate device memory
    int8_t *d_input;
    float *d_output;
    hipMalloc(&d_input, size * sizeof(int8_t));
    hipMalloc(&d_output, size * size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);
    diagflat_kernel_int8<<<numBlocks, threadsPerBlock>>>(d_input, d_output, size);

    // Copy result back to host
    hipMemcpy(output, d_output, size * size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

} // extern "C"
