#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end
#include <cutlass/cutlass.h>

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for chain matrix multiplication and ReLU using bfloat16
__global__ void chain_matmul_relu_kernel_bf16(const float* input_tensor, const float* weight1, const float* weight2, float* output, 
                                        int m, int n, int k1, int k2) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k1; ++i) {
            for (int j = 0; j < k2; ++j) {
                __hip_bfloat16 a = float_to_bfloat16(input_tensor[row * k1 + i]);
                __hip_bfloat16 b = float_to_bfloat16(weight1[col * k1 + i]);
                __hip_bfloat16 c = float_to_bfloat16(weight2[j * k2 + i]); 
                sum += bfloat16_to_float(__hmul(__hmul(a, b), c)); 
            }
        }
        output[row * n + col] = fmaxf(sum, 0.0f);  // ReLU activation
    }
}

extern "C" {

void chain_matmul_bfloat16_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    const float* weight1 = va_arg(args, const float*);
    int weight1_dim0 = va_arg(args, int);
    int weight1_dim1 = va_arg(args, int);

    const float* weight2 = va_arg(args, const float*);
    int weight2_dim0 = va_arg(args, int);
    int weight2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int hidden_dim = weight1_dim0;
    int output_dim = weight2_dim0;

    // Allocate device memory
    float *d_input, *d_weight1, *d_weight2, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight1, hidden_dim * input_dim * sizeof(float));
    hipMalloc(&d_weight2, output_dim * hidden_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight1, weight1, hidden_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight2, weight2, output_dim * hidden_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    chain_matmul_relu_kernel_bf16<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight1, d_weight2, d_output, batch_size, output_dim, input_dim, hidden_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight1);
    hipFree(d_weight2);
    hipFree(d_output);
}

}  // extern "C"
