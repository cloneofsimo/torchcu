
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __half
__device__ __forceinline__ __half float_to_half(float f) {
    return __float2half_rn(f);
}

// CUDA kernel for calculating Hamming distance
__global__ void pairwise_hamming_distance_kernel(const int8_t* x, const int8_t* y, int* distance, int seq_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < seq_len && j < seq_len) {
        distance[i * seq_len + j] = __popc(x[i] ^ y[j]);
    }
}

// CUDA kernel for relative positional encoding
__global__ void relative_positional_encoding_kernel(const int* distance, __half* relative_position_embeddings, 
                                                    int batch_size, int seq_len, int max_relative_positions) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < seq_len && j < seq_len) {
        int dist = distance[i * seq_len + j];
        int index = dist + max_relative_positions;
        relative_position_embeddings[i * seq_len + j] = float_to_half(index < 2 * max_relative_positions + 1 ? 1.0f : 0.0f);
    }
}

extern "C" {

void relative_positional_encoding_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const int8_t* input_tensor = va_arg(args, const int8_t*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract seq_len
    int seq_len = va_arg(args, int);

    // Extract max_relative_positions
    int max_relative_positions = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    __half* output = va_arg(args, __half*);

    va_end(args);

    // Allocate device memory
    int* d_distance;
    hipMalloc(&d_distance, seq_len * seq_len * sizeof(int));
    hipMalloc(&output, seq_len * seq_len * sizeof(__half));

    // Calculate pairwise Hamming distances on the device
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((seq_len + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (seq_len + threadsPerBlock.y - 1) / threadsPerBlock.y);
    pairwise_hamming_distance_kernel<<<numBlocks, threadsPerBlock>>>(input_tensor, input_tensor, d_distance, seq_len);

    // Compute relative positional encodings on the device
    relative_positional_encoding_kernel<<<numBlocks, threadsPerBlock>>>(d_distance, output, 1, seq_len, max_relative_positions);

    // Free device memory
    hipFree(d_distance);
}

}  // extern "C"
