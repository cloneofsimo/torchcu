
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdarg.h> 

// CUDA kernel for flattened einsum summation with fp16
__global__ void einsum_kernel_fp16(const half* input_tensor, const half* weight, half* output,
                                        int batch_size, int input_dim, int output_dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < batch_size && j < output_dim) {
        half sum = __int2half_rn(0);
        for (int k = 0; k < input_dim; ++k) {
            sum = __hadd(sum, __hmul(input_tensor[i * input_dim + k], weight[k * output_dim + j]));
        }
        output[i * output_dim + j] = sum;
    }
}

extern "C" {

void flatten_einsum_fp16_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    half* output = va_arg(args, half*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1 * input_tensor_dim2;
    int output_dim = weight_dim1;

    // Allocate device memory
    half *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(half));
    hipMalloc(&d_weight, weight_dim0 * weight_dim1 * sizeof(half));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(half));

    // Copy input data to device (converting to fp16)
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (output_dim + threadsPerBlock.y - 1) / threadsPerBlock.y);

    einsum_kernel_fp16<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, batch_size, input_dim, output_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(half), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
