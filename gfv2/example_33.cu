
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for matrix multiplication and ReLU using bfloat16
__global__ void matmul_bf16_int8_kernel(const float* input_tensor, const float* weight, const float* bias, int8_t* output,
                                        int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            __hip_bfloat16 a = float_to_bfloat16(input_tensor[row * k + i]);
            __hip_bfloat16 b = float_to_bfloat16(weight[col * k + i]);  // Transposed access
            sum += bfloat16_to_float(__hmul(a, b));
        }
        if (bias != nullptr) {
            sum += bias[col];
        }
        output[row * n + col] = (int8_t)sum;
    }
}

extern "C" {

void int8_linear_bf16_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract bias tensor (optional)
    const float* bias = va_arg(args, const float*);

    // Extract output tensor (assuming it's preallocated)
    int8_t* output = va_arg(args, int8_t*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_bias;
    int8_t *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    if (bias != nullptr) {
        hipMalloc(&d_bias, output_dim * sizeof(float));
    }
    hipMalloc(&d_output, batch_size * output_dim * sizeof(int8_t));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    if (bias != nullptr) {
        hipMemcpy(d_bias, bias, output_dim * sizeof(float), hipMemcpyHostToDevice);
    }

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_bf16_int8_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_bias, d_output, batch_size, output_dim, input_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    if (bias != nullptr) {
        hipFree(d_bias);
    }
    hipFree(d_output);
}

}  // extern "C"
