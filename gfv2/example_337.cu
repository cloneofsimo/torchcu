
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdarg.h> 

// CUDA kernel for adaptive softmax
__global__ void adaptive_softmax_kernel(const float* input_tensor, const float* weight, float* output, 
                                        int batch_size, int input_dim, int output_dim, float layer_scaling) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batch_size && col < output_dim) {
        float sum = 0.0f;
        for (int i = 0; i < input_dim; ++i) {
            sum += input_tensor[row * input_dim + i] * weight[col * input_dim + i] * layer_scaling;
        }
        // Apply softmax
        output[row * output_dim + col] = expf(sum); // Calculate exp
        // For normalization
        for (int i = 0; i < output_dim; ++i) {
            output[row * output_dim + i] += expf(sum); // Calculate exp
        }
        output[row * output_dim + col] = logf(output[row * output_dim + col]); // Take log
    }
}

extern "C" {

void adaptive_softmax_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract layer scaling
    float layer_scaling = va_arg(args, double);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    adaptive_softmax_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, batch_size, input_dim, output_dim, layer_scaling
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
