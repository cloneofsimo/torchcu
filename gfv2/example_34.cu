#include "hip/hip_runtime.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>

// Function to convert float to half
__device__ __forceinline__ half float_to_half(float f) {
    return __float2half_rn(f);
}

// Function to convert half to float
__device__ __forceinline__ float half_to_float(half h) {
    return __half2float(h);
}

extern "C" {

void image_jacobian_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);
    int weight_dim2 = va_arg(args, int);
    int weight_dim3 = va_arg(args, int);

    // Extract bias tensor
    const float* bias = va_arg(args, const float*);
    int bias_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // --- CUDNN setup ---
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // --- Create CUDNN tensors ---
    hipdnnTensorDescriptor_t inputTensorDesc, weightTensorDesc, outputTensorDesc;
    hipdnnCreateTensorDescriptor(&inputTensorDesc);
    hipdnnCreateTensorDescriptor(&weightTensorDesc);
    hipdnnCreateTensorDescriptor(&outputTensorDesc);

    // --- Set tensor descriptors ---
    hipdnnSetTensorNdDescriptor(inputTensorDesc, HIPDNN_DATA_FLOAT, 4, 
                                &input_tensor_dim0, &input_tensor_dim1, 
                                &input_tensor_dim2, &input_tensor_dim3);
    hipdnnSetTensorNdDescriptor(weightTensorDesc, HIPDNN_DATA_FLOAT, 4,
                                &weight_dim0, &weight_dim1, 
                                &weight_dim2, &weight_dim3);
    hipdnnSetTensorNdDescriptor(outputTensorDesc, HIPDNN_DATA_FLOAT, 4, 
                                &input_tensor_dim0, &weight_dim0,
                                &input_tensor_dim2, &input_tensor_dim3);

    // --- Create convolution descriptor ---
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolutionNdDescriptor(convDesc, 2, // Number of spatial dimensions
                                    {weight_dim2, weight_dim3}, 
                                    {0, 0}, {0, 0}, 
                                    CUDNN_CONVOLUTION_CROSS_CORRELATION, 
                                    HIPDNN_DATA_FLOAT); // Data type

    // --- Allocate device memory ---
    float *d_input, *d_weight, *d_bias, *d_output;
    hipMalloc(&d_input, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float));
    hipMalloc(&d_bias, bias_dim0 * sizeof(float));
    hipMalloc(&d_output, input_tensor_dim0 * weight_dim0 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));

    // --- Copy data to device ---
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, bias_dim0 * sizeof(float), hipMemcpyHostToDevice);

    // --- Perform convolution with CUDNN ---
    hipdnnConvolutionForward(cudnnHandle, // CUDNN handle
                            &alpha, // Alpha value for scaling output
                            inputTensorDesc, // Input tensor descriptor
                            d_input, // Input tensor data
                            weightTensorDesc, // Weight tensor descriptor
                            d_weight, // Weight tensor data
                            convDesc, // Convolution descriptor
                            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, // Algorithm
                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, // Preference for fastest algorithm
                            &beta, // Beta value for scaling output
                            outputTensorDesc, // Output tensor descriptor
                            d_output // Output tensor data
                            );

    // --- Apply bias ---
    hipdnnAddTensor(cudnnHandle, // CUDNN handle
                   &alpha, // Alpha value for scaling bias
                   biasTensorDesc, // Bias tensor descriptor
                   d_bias, // Bias tensor data
                   &beta, // Beta value for scaling output
                   outputTensorDesc, // Output tensor descriptor
                   d_output // Output tensor data
                   );

    // --- Copy result back to host ---
    hipMemcpy(output, d_output, input_tensor_dim0 * weight_dim0 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // --- Free CUDNN resources ---
    hipdnnDestroyTensorDescriptor(inputTensorDesc);
    hipdnnDestroyTensorDescriptor(weightTensorDesc);
    hipdnnDestroyTensorDescriptor(outputTensorDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroy(cudnnHandle);

    // --- Free device memory ---
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);
}
} // extern "C"
