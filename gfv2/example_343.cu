
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __half
__device__ __forceinline__ __half float_to_half(float f) {
    return __float2half_rn(f);  // Round-to-nearest even for better precision
}

// Helper function to convert __half to float
__device__ __forceinline__ float half_to_float(__half h) {
    return __half2float(h);
}

__global__ void elementwise_min_kernel_fp16(const float* input1, const float* input2, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        __half a = float_to_half(input1[idx]);
        __half b = float_to_half(input2[idx]);
        output[idx] = half_to_float(fminf(a, b));  // Element-wise minimum
    }
}

extern "C" {

void elementwise_min_fp16(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input1 = va_arg(args, const float*);
    int input1_dim0 = va_arg(args, int);
    int input1_dim1 = va_arg(args, int);

    const float* input2 = va_arg(args, const float*);
    int input2_dim0 = va_arg(args, int);
    int input2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input1_dim0 * input1_dim1;

    // Allocate device memory
    float *d_input1, *d_input2, *d_output;
    hipMalloc(&d_input1, size * sizeof(float));
    hipMalloc(&d_input2, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input1, input1, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    elementwise_min_kernel_fp16<<<numBlocks, threadsPerBlock>>>(
        d_input1, d_input2, d_output, size
    );

    // Copy result back to host
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
}

}  // extern "C"
