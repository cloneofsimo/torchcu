#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// CUDA kernel for permute, unfold, and convolution
__global__ void conv_permute_unfold_kernel(const float* input_tensor, const float* weight, const float* bias, 
                                        float* output, int batch_size, int in_channels, int in_height, int in_width,
                                        int out_channels, int kernel_height, int kernel_width) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.z * blockDim.z + threadIdx.z;
    int o = threadIdx.w;

    if (b < batch_size && h < in_height && w < in_width && o < out_channels) {
        float sum = 0.0f;
        for (int kh = 0; kh < kernel_height; ++kh) {
            for (int kw = 0; kw < kernel_width; ++kw) {
                int in_idx = (b * in_channels * in_height * in_width) + (o * in_height * in_width) + 
                            ((h + kh) * in_width) + (w + kw);
                int weight_idx = (o * in_channels * kernel_height * kernel_width) + 
                                 (kh * kernel_width) + kw;
                sum += input_tensor[in_idx] * weight[weight_idx];
            }
        }

        // Add bias
        sum += bias[o];
        output[(b * out_channels * in_height * in_width) + (h * in_width * out_channels) + 
                 (w * out_channels) + o] = sum;
    }
}

extern "C" {

void conv_permute_unfold(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);
    int weight_dim2 = va_arg(args, int);
    int weight_dim3 = va_arg(args, int);

    // Extract bias tensor
    const float* bias = va_arg(args, const float*);
    int bias_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int in_channels = input_tensor_dim1;
    int in_height = input_tensor_dim2;
    int in_width = input_tensor_dim3;

    int out_channels = weight_dim0;
    int kernel_height = weight_dim2;
    int kernel_width = weight_dim3;

    // Allocate device memory
    float *d_input, *d_weight, *d_bias, *d_output;
    hipMalloc(&d_input, batch_size * in_channels * in_height * in_width * sizeof(float));
    hipMalloc(&d_weight, out_channels * in_channels * kernel_height * kernel_width * sizeof(float));
    hipMalloc(&d_bias, out_channels * sizeof(float));
    hipMalloc(&d_output, batch_size * out_channels * in_height * in_width * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * in_channels * in_height * in_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, out_channels * in_channels * kernel_height * kernel_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, out_channels * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16, 16, 1);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (in_height + threadsPerBlock.y - 1) / threadsPerBlock.y, 
                   (in_width + threadsPerBlock.z - 1) / threadsPerBlock.z);

    conv_permute_unfold_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_bias, d_output,
        batch_size, in_channels, in_height, in_width,
        out_channels, kernel_height, kernel_width
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * out_channels * in_height * in_width * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);
}

}  // extern "C"
