
#include <hip/hip_runtime.h>

#include <stdarg.h>

// CUDA kernel for the complex function
__global__ void complex_function_kernel(const float* input_tensor, const float* weight1, const float* weight2, float* output,
                                        int batch_size, int input_dim, int hidden_dim1, int hidden_dim2) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batch_size && col < hidden_dim2) {
        float sum1 = 0.0f;
        for (int i = 0; i < input_dim; ++i) {
            sum1 += input_tensor[row * input_dim + i] * weight1[col * input_dim + i];
        }

        float sum2 = 0.0f;
        for (int i = 0; i < hidden_dim1; ++i) {
            sum2 += sum1 * weight2[col * hidden_dim1 + i];
        }

        output[row * hidden_dim2 + col] = sum1 + sum2;
    }
}

extern "C" {

void my_complex_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight1 tensor
    const float* weight1 = va_arg(args, const float*);
    int weight1_dim0 = va_arg(args, int);
    int weight1_dim1 = va_arg(args, int);

    // Extract weight2 tensor
    const float* weight2 = va_arg(args, const float*);
    int weight2_dim0 = va_arg(args, int);
    int weight2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int hidden_dim1 = weight1_dim0;
    int hidden_dim2 = weight2_dim0;

    // Allocate device memory
    float *d_input, *d_weight1, *d_weight2, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight1, hidden_dim1 * input_dim * sizeof(float));
    hipMalloc(&d_weight2, hidden_dim2 * hidden_dim1 * sizeof(float));
    hipMalloc(&d_output, batch_size * hidden_dim2 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight1, weight1, hidden_dim1 * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight2, weight2, hidden_dim2 * hidden_dim1 * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 16);
    dim3 numBlocks((hidden_dim2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    complex_function_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight1, d_weight2, d_output, batch_size, input_dim, hidden_dim1, hidden_dim2
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * hidden_dim2 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight1);
    hipFree(d_weight2);
    hipFree(d_output);
}

}  // extern "C"
