
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdarg.h>

// CUDA kernel for multi-margin loss with constant padding
__global__ void multi_margin_loss_kernel(const float* input_tensor, const float* weight, float* output,
                                        int target, int padding, float pad_value, int input_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Handle padding
    if (idx >= padding && idx < input_size + padding) {
        float sum = 0.0f;
        for (int i = 0; i < input_size; ++i) {
            float val = input_tensor[idx + i];
            float w = weight[i];
            sum += fmaxf(0.0f, w * (val - input_tensor[idx + target] + 1.0f));
        }
        output[0] = sum;
    } else if (idx < padding || idx >= input_size + padding) {
        output[0] = pad_value;
    }
}

extern "C" {

void multi_margin_loss_with_padding(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);

    // Extract target
    int target = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);

    // Extract padding and pad_value
    int padding = va_arg(args, int);
    float pad_value = va_arg(args, float);

    va_end(args);

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, input_tensor_dim0 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((input_tensor_dim0 + 2 * padding + threadsPerBlock.x - 1) / threadsPerBlock.x);

    multi_margin_loss_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, target, padding, pad_value, input_tensor_dim0
    );

    // Copy result back to host
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

} // extern "C"
