
#include <hip/hip_runtime.h>
#include <hipDNN.h>

extern "C" {

void elementwise_min_cudnn(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input_tensor1 = va_arg(args, const float*);
    int input_tensor1_dim0 = va_arg(args, int);
    int input_tensor1_dim1 = va_arg(args, int);
    const float* input_tensor2 = va_arg(args, const float*);
    int input_tensor2_dim0 = va_arg(args, int);
    int input_tensor2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // Check input tensor shapes
    if (input_tensor1_dim0 != input_tensor2_dim0 ||
        input_tensor1_dim1 != input_tensor2_dim1) {
        printf("Error: Input tensors have incompatible shapes.\n");
        return;
    }

    // Initialize cuDNN
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // Allocate device memory
    float* d_input1, *d_input2, *d_output;
    hipMalloc(&d_input1, input_tensor1_dim0 * input_tensor1_dim1 * sizeof(float));
    hipMalloc(&d_input2, input_tensor2_dim0 * input_tensor2_dim1 * sizeof(float));
    hipMalloc(&d_output, input_tensor1_dim0 * input_tensor1_dim1 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input1, input_tensor1, input_tensor1_dim0 * input_tensor1_dim1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input_tensor2, input_tensor2_dim0 * input_tensor2_dim1 * sizeof(float), hipMemcpyHostToDevice);

    // Create cuDNN tensors
    hipdnnTensorDescriptor_t input_tensor_desc1, input_tensor_desc2, output_tensor_desc;
    hipdnnCreateTensorDescriptor(&input_tensor_desc1);
    hipdnnCreateTensorDescriptor(&input_tensor_desc2);
    hipdnnCreateTensorDescriptor(&output_tensor_desc);

    // Set tensor descriptors
    hipdnnSetTensor4dDescriptor(input_tensor_desc1, HIPDNN_DATA_FLOAT, 1, 1, input_tensor1_dim0, input_tensor1_dim1);
    hipdnnSetTensor4dDescriptor(input_tensor_desc2, HIPDNN_DATA_FLOAT, 1, 1, input_tensor2_dim0, input_tensor2_dim1);
    hipdnnSetTensor4dDescriptor(output_tensor_desc, HIPDNN_DATA_FLOAT, 1, 1, input_tensor1_dim0, input_tensor1_dim1);

    // cuDNN operation
    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, CUDNN_ACTIVATION_MIN, HIPDNN_PROPAGATE_NAN, 0.0f);

    hipdnnActivationForward(cudnnHandle, actDesc, d_input1, input_tensor_desc1, d_input2, input_tensor_desc2, d_output, output_tensor_desc);

    // Copy result back to host
    hipMemcpy(output, d_output, input_tensor1_dim0 * input_tensor1_dim1 * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipdnnDestroyTensorDescriptor(input_tensor_desc1);
    hipdnnDestroyTensorDescriptor(input_tensor_desc2);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);
    hipdnnDestroyActivationDescriptor(actDesc);
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
    hipdnnDestroy(cudnnHandle);
}

} // extern "C"
