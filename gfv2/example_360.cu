#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdarg.h> 

// CUDA kernel for matrix multiplication, fused dropout, and tanh activation
__global__ void matmul_dropout_tanh_kernel(const float* input_tensor, const float* weight, float* output, 
                                        int m, int n, int k, float dropout_p, int seed) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += input_tensor[row * k + i] * weight[col * k + i];  // Transposed access
        }
        // Fused dropout
        float r = (float)rand_r(&seed) / RAND_MAX;
        if (r < dropout_p) {
            output[row * n + col] = 0.0f;
        } else {
            output[row * n + col] = tanhf(sum);  // tanh activation
        }
    }
}

extern "C" {

void my_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract dropout probability
    float dropout_p = va_arg(args, double);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Generate a random seed for each block to ensure independent dropout for each block
    int seed = 0;  // Seed for random number generator
    int *d_seed;
    hipMalloc(&d_seed, sizeof(int));
    hipMemcpy(d_seed, &seed, sizeof(int), hipMemcpyHostToDevice);

    matmul_dropout_tanh_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, batch_size, output_dim, input_dim, dropout_p, *d_seed
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    hipFree(d_seed);
}

}  // extern "C"
