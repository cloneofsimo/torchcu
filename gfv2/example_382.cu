
#include <hip/hip_runtime.h>

#include <stdarg.h> 

__global__ void my_complex_function_kernel(const float* input_tensor, const float* weight, float* output,
                                        int batch_size, int input_dim, int output_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < batch_size) {
        float sum = 0.0f;
        for (int i = 0; i < input_dim; ++i) {
            sum += input_tensor[idx * input_dim + i] * weight[i];
        }
        output[idx] = fmaxf(sum * 2.0f, 0.0f); 
    }
}

extern "C" {

void my_complex_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (batch_size + threadsPerBlock - 1) / threadsPerBlock;
    my_complex_function_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, batch_size, input_dim, output_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}

}  // extern "C"
