#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdarg.h> 

__global__ void bucketize_invert_tanh_kernel(const float* input_tensor, const float* buckets, 
                                            half* output_tensor, int* bucketized_tensor, 
                                            int input_size, int bucket_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < input_size) {
        int bucket_idx = 0;
        for (int i = 0; i < bucket_size; ++i) {
            if (input_tensor[idx] <= buckets[i]) {
                bucket_idx = i;
                break;
            }
        }
        bucketized_tensor[idx] = bucket_idx;
        
        // Invert and apply tanh
        output_tensor[idx] = __float2half_rn(tanhf(1.0f - (float)bucket_idx));
    }
}

extern "C" {

void my_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_size = va_arg(args, int);

    // Extract buckets tensor
    const float* buckets = va_arg(args, const float*);
    int bucket_size = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    half* output_tensor = va_arg(args, half*);
    int* bucketized_tensor = va_arg(args, int*);

    va_end(args);

    // Allocate device memory
    float *d_input, *d_buckets;
    half *d_output_tensor;
    int *d_bucketized_tensor;
    hipMalloc(&d_input, input_tensor_size * sizeof(float));
    hipMalloc(&d_buckets, bucket_size * sizeof(float));
    hipMalloc(&d_output_tensor, input_tensor_size * sizeof(half));
    hipMalloc(&d_bucketized_tensor, input_tensor_size * sizeof(int));

    // Copy data to device
    hipMemcpy(d_input, input_tensor, input_tensor_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_buckets, buckets, bucket_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256; // Adjust this value based on your GPU
    int numBlocks = (input_tensor_size + threadsPerBlock - 1) / threadsPerBlock;

    bucketize_invert_tanh_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_buckets, d_output_tensor, d_bucketized_tensor, input_tensor_size, bucket_size
    );

    // Copy result back to host
    hipMemcpy(output_tensor, d_output_tensor, input_tensor_size * sizeof(half), hipMemcpyDeviceToHost);
    hipMemcpy(bucketized_tensor, d_bucketized_tensor, input_tensor_size * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_buckets);
    hipFree(d_output_tensor);
    hipFree(d_bucketized_tensor);
}

}  // extern "C"
