
#include <hip/hip_runtime.h>

#include <stdarg.h> 

// CUDA kernel for pairwise Hamming distance and addition
__global__ void pairwise_hamming_distance_add_kernel(const float* input_tensor, const float* weights, float* output, 
                                        int batch_size, int feature_dim) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < batch_size && col < feature_dim) {
        int hamming_distance = 0;
        for (int i = 0; i < 8; ++i) { // Assuming 8-bit integer representation
            int input_val = ((int)input_tensor[row * feature_dim + col] >> i) & 1;
            int weight_val = ((int)weights[col] >> i) & 1;
            if (input_val != weight_val) {
                hamming_distance++;
            }
        }
        output[row * feature_dim + col] = input_tensor[row * feature_dim + col] + hamming_distance;
    }
}

extern "C" {

void pairwise_hamming_distance_add(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weights tensor
    const float* weights = va_arg(args, const float*);
    int weights_dim0 = va_arg(args, int); 

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int feature_dim = input_tensor_dim1;

    // Allocate device memory
    float *d_input, *d_weights, *d_output;
    hipMalloc(&d_input, batch_size * feature_dim * sizeof(float));
    hipMalloc(&d_weights, feature_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * feature_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * feature_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, feature_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((feature_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    pairwise_hamming_distance_add_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weights, d_output, batch_size, feature_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * feature_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_output);
}

}  // extern "C"
