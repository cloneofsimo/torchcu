#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <stdarg.h> 
#include <stdio.h>

// CUDA kernel for simple data processing
__global__ void process_data_kernel(const float* input_tensor, float* output, float sum_data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input_tensor[idx] * sum_data;
    }
}

// Helper function to load data from a file
float load_data_sum(const char* filename) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Error: Could not open file %s\n", filename);
        return 0.0f;
    }

    float sum = 0.0f;
    float data_value;
    while (fread(&data_value, sizeof(float), 1, fp) == 1) {
        sum += data_value;
    }

    fclose(fp);
    return sum;
}

extern "C" {

void process_data(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int); 

    // Extract filename (dummy type for now)
    const char* filename = va_arg(args, const char*);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input_tensor_dim0;

    // Load data from file and calculate sum
    float sum_data = load_data_sum(filename);

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    process_data_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, sum_data, size);

    // Copy result back to host (convert to fp16)
    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < size; ++i) {
        output[i] = __float2half_rn(output[i]); // Convert to fp16
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
