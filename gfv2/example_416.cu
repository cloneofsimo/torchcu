
#include <hip/hip_runtime.h>

#include <stdarg.h>

// CUDA kernel for the complex operation with layer scaling
__global__ void complex_kernel(const float* input_tensor, const float* weight, const float* bias, float scale, 
                              float* output, float* pre_activation,
                              int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += input_tensor[row * k + i] * weight[col * k + i]; 
        }
        sum += bias[col]; // Add bias
        pre_activation[row * n + col] = sum; // Store pre-activation
        output[row * n + col] = fmaxf(sum, 0.0f) * scale; // ReLU and scaling
    }
}

extern "C" {

void my_complex_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract bias tensor
    const float* bias = va_arg(args, const float*);
    int bias_dim0 = va_arg(args, int);

    // Extract scale value
    float scale = va_arg(args, double); 

    // Extract output tensors (assuming they're preallocated)
    float* output = va_arg(args, float*);
    float* pre_activation = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_bias, *d_output, *d_pre_activation;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_bias, output_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));
    hipMalloc(&d_pre_activation, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, output_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    complex_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_bias, scale, d_output, d_pre_activation, 
        batch_size, output_dim, input_dim
    );

    // Copy results back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(pre_activation, d_pre_activation, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);
    hipFree(d_pre_activation);
}

} // extern "C"
