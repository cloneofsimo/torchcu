#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <stdarg.h>  // Add this for va_list, va_start, va_end

// Helper function to convert float to __hip_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __hip_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

// CUDA kernel for element-wise power, matrix multiplication, and bfloat16 conversion
__global__ void power_matmul_kernel_bf16(const float* input_tensor, const float* weight, 
                                        __hip_bfloat16* output, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            float input_val = input_tensor[row * k + i];
            sum += input_val * input_val * weight[col * k + i];  // Transposed access
        }
        output[row * n + col] = float_to_bfloat16(sum); 
    }
}

extern "C" {

void my_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    __hip_bfloat16* output = va_arg(args, __hip_bfloat16*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight;
    __hip_bfloat16 *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(__hip_bfloat16));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    power_matmul_kernel_bf16<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_output, batch_size, output_dim, input_dim
    );

    // Copy result back to host (converting from bfloat16 to float)
    float* output_float = new float[batch_size * output_dim];
    hipMemcpy(output_float, d_output, batch_size * output_dim * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost);
    for (int i = 0; i < batch_size * output_dim; ++i) {
        output[i] = float_to_bfloat16(output_float[i]); 
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    delete[] output_float;
}

}  // extern "C"
