
#include <hip/hip_runtime.h>

#include <stdarg.h>

// CUDA kernel for pruned linear transformation and int8 quantization
__global__ void pruned_linear_int8_kernel(const float* input_tensor, const float* weight, 
                                         const bool* pruning_mask, int8_t* output,
                                         int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            if (pruning_mask[col * k + i]) { // Apply pruning mask
                sum += input_tensor[row * k + i] * weight[col * k + i];
            }
        }
        output[row * n + col] = (int8_t)sum; // Quantize to int8
    }
}

extern "C" {

void pruned_linear_int8(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract pruning mask
    const bool* pruning_mask = va_arg(args, const bool*);

    // Extract output tensor (assuming it's preallocated)
    int8_t* output = va_arg(args, int8_t*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight;
    bool *d_pruning_mask;
    int8_t *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_pruning_mask, output_dim * input_dim * sizeof(bool));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(int8_t));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pruning_mask, pruning_mask, output_dim * input_dim * sizeof(bool), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(32, 32); // Adjust block size for optimal performance
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    pruned_linear_int8_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_pruning_mask, d_output, batch_size, output_dim, input_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_pruning_mask);
    hipFree(d_output);
}

}  // extern "C"
