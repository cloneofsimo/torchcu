
#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>
#include <stdarg.h>

#define SELU_ALPHA 1.6732632423543772848170429916717
#define SELU_SCALE 1.0507009873554804934193349852946

__device__ float selu(float x) {
    return (x > 0.0f) ? SELU_SCALE * x : SELU_SCALE * SELU_ALPHA * (expf(x) - 1.0f);
}

__global__ void my_function_kernel(const float* input, const float* weight, const float* bias, float* output, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += input[row * k + i] * weight[col * k + i];
        }
        sum += bias[col];
        output[row * n + col] = selu(sum);
    }
}

__global__ void adaptive_max_pool_kernel(const float* input, float* output, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float max_val = input[row * n + col];
        for (int i = 1; i < n; ++i) {
            max_val = fmaxf(max_val, input[row * n + col + i]);
        }
        output[row] = max_val;
    }
}

extern "C" {
    void my_function(int num_args, ...) {
        va_list args;
        va_start(args, num_args);

        // Extract input tensor
        const float* input = va_arg(args, const float*);
        int input_dim0 = va_arg(args, int);
        int input_dim1 = va_arg(args, int);

        // Extract weight tensor
        const float* weight = va_arg(args, const float*);
        int weight_dim0 = va_arg(args, int);
        int weight_dim1 = va_arg(args, int);

        // Extract bias tensor
        const float* bias = va_arg(args, const float*);
        int bias_dim = va_arg(args, int);

        // Extract output tensor (assuming it's preallocated)
        float* output = va_arg(args, float*);

        va_end(args);

        int m = input_dim0;
        int k = input_dim1;
        int n = weight_dim0;

        // Allocate device memory
        float *d_input, *d_weight, *d_bias, *d_output, *d_output_pool;
        hipMalloc(&d_input, m * k * sizeof(float));
        hipMalloc(&d_weight, n * k * sizeof(float));
        hipMalloc(&d_bias, n * sizeof(float));
        hipMalloc(&d_output, m * n * sizeof(float));
        hipMalloc(&d_output_pool, m * sizeof(float));

        // Copy input data to device
        hipMemcpy(d_input, input, m * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_weight, weight, n * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_bias, bias, n * sizeof(float), hipMemcpyHostToDevice);

        // Launch kernel for matrix multiplication and SELU activation
        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);
        my_function_kernel<<<numBlocks, threadsPerBlock>>>(d_input, d_weight, d_bias, d_output, m, n, k);

        // Launch kernel for adaptive max pooling
        numBlocks = (m + threadsPerBlock.x - 1) / threadsPerBlock.x;
        adaptive_max_pool_kernel<<<numBlocks, threadsPerBlock>>>(d_output, d_output_pool, m, n);

        // Perform inplace division on the pooled output
        hipMemcpy(d_output, d_output_pool, m * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(output, d_output, m * sizeof(float), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_input);
        hipFree(d_weight);
        hipFree(d_bias);
        hipFree(d_output);
        hipFree(d_output_pool);
    }
}
