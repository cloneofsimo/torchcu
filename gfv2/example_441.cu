
#include <hip/hip_runtime.h>

#include <stdarg.h> 

// CUDA kernel for baddbmm and masked selection
__global__ void baddbmm_masked_kernel(const float* input_tensor, const float* weight, const bool* mask, float* output, 
                                        int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = input_tensor[row * n + col]; // Initialize with input value
        for (int i = 0; i < k; ++i) {
            sum += input_tensor[row * k + i] * weight[col * k + i];
        }
        output[row * n + col] = sum;

        // Apply mask
        if (!mask[row * n + col]) {
            output[row * n + col] = 0.0f;
        }
    }
}

// CUDA kernel for reduction (sum)
__global__ void reduction_kernel(const float* output, float* result, int m, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < m * n) {
        atomicAdd(result, output[idx]);
    }
}

extern "C" {

void complex_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract mask tensor
    const bool* mask = va_arg(args, const bool*);
    int mask_dim0 = va_arg(args, int);
    int mask_dim1 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weight_dim0;

    // Allocate device memory
    float *d_input, *d_weight, *d_output;
    bool *d_mask;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weight, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));
    hipMalloc(&d_mask, batch_size * input_dim * sizeof(bool));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, batch_size * input_dim * sizeof(bool), hipMemcpyHostToDevice);

    // Launch baddbmm and masked selection kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    baddbmm_masked_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weight, d_mask, d_output, batch_size, output_dim, input_dim
    );

    // Allocate device memory for result
    float *d_result;
    hipMalloc(&d_result, sizeof(float));

    // Launch reduction kernel
    numBlocks = (batch_size * output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x;

    reduction_kernel<<<numBlocks, threadsPerBlock>>>(d_output, d_result, batch_size, output_dim);

    // Copy result back to host
    hipMemcpy(output, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    hipFree(d_mask);
    hipFree(d_result);
}

}  // extern "C"
