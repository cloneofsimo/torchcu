#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

// CUDA kernel for cross-entropy with bucketized weights
__global__ void cross_entropy_kernel(const float* input, const int* target, const float* weights, 
                                     const float* bucket_boundaries, float* loss, int batch_size, int num_classes, int num_buckets) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        int target_class = target[idx];
        float sum_exp = 0.0f;
        for (int c = 0; c < num_classes; ++c) {
            sum_exp += expf(input[idx * num_classes + c]);
        }
        float log_sum_exp = logf(sum_exp);
        float input_val = input[idx * num_classes + target_class];
        float loss_val = log_sum_exp - input_val;

        // Bucketize weights on the device
        int bucket_idx = 0;
        while (bucket_idx < num_buckets - 1 && weights[target_class] > bucket_boundaries[bucket_idx]) {
            bucket_idx++;
        }
        loss[idx] = loss_val * bucket_idx; 
    }
}

extern "C" {

void cross_entropy_with_bucketized_weights(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    const float* input = va_arg(args, const float*);
    int batch_size = va_arg(args, int);
    int num_classes = va_arg(args, int);

    const int* target = va_arg(args, const int*);

    const float* weights = va_arg(args, const float*);

    int num_buckets = va_arg(args, int);

    float* output = va_arg(args, float*);

    va_end(args);

    // Allocate device memory
    float *d_input, *d_weights, *d_loss, *d_bucket_boundaries;
    int *d_target;
    hipMalloc(&d_input, batch_size * num_classes * sizeof(float));
    hipMalloc(&d_weights, num_classes * sizeof(float));
    hipMalloc(&d_loss, batch_size * sizeof(float));
    hipMalloc(&d_target, batch_size * sizeof(int));
    hipMalloc(&d_bucket_boundaries, (num_buckets - 1) * sizeof(float)); 

    // Copy input data to device
    hipMemcpy(d_input, input, batch_size * num_classes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_target, target, batch_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, num_classes * sizeof(float), hipMemcpyHostToDevice);

    // Generate bucket boundaries on the device
    float* bucket_boundaries = new float[num_buckets - 1]; 
    for (int i = 0; i < num_buckets - 1; i++) {
        bucket_boundaries[i] = weights[0] + (weights[num_classes - 1] - weights[0]) * (i + 1.0f) / num_buckets;
    }
    hipMemcpy(d_bucket_boundaries, bucket_boundaries, (num_buckets - 1) * sizeof(float), hipMemcpyHostToDevice);
    delete[] bucket_boundaries;

    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (batch_size + threadsPerBlock - 1) / threadsPerBlock;
    cross_entropy_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_target, d_weights, d_bucket_boundaries, d_loss, batch_size, num_classes, num_buckets
    );

    // Copy result back to host
    hipMemcpy(output, d_loss, batch_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_loss);
    hipFree(d_target);
    hipFree(d_bucket_boundaries);
}

}  // extern "C"
