
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp16.h>
#include <hipDNN.h>

extern "C" {

void depthwise_conv2d_relu_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);
    int input_tensor_dim2 = va_arg(args, int);
    int input_tensor_dim3 = va_arg(args, int);

    // Extract weight tensor
    const float* weight = va_arg(args, const float*);
    int weight_dim0 = va_arg(args, int);
    int weight_dim1 = va_arg(args, int);
    int weight_dim2 = va_arg(args, int);
    int weight_dim3 = va_arg(args, int);

    // Extract bias tensor
    const float* bias = va_arg(args, const float*);
    int bias_dim0 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    // CUDA Initialization
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);

    // Create cudnnTensorDescriptor
    hipdnnTensorDescriptor_t inputDesc, weightDesc, biasDesc, outputDesc;
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&weightDesc);
    hipdnnCreateTensorDescriptor(&biasDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);

    // Set Tensor Descriptors
    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);
    hipdnnSetTensor4dDescriptor(weightDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                weight_dim0, weight_dim1, weight_dim2, weight_dim3);
    cudnnSetTensor1dDescriptor(biasDesc, HIPDNN_DATA_FLOAT, bias_dim0);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                input_tensor_dim0, input_tensor_dim1, input_tensor_dim2, input_tensor_dim3);

    // Create cudnnFilterDescriptor
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnCreateFilterDescriptor(&filterDesc);
    hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                weight_dim0, weight_dim1, weight_dim2, weight_dim3);

    // Create cudnnConvolutionDescriptor
    hipdnnConvolutionDescriptor_t convDesc;
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc, 1, 1, 1, 1, 1, 1, CUDNN_CONVOLUTION_CROSS_CORRELATION);

    // Allocate device memory
    float *d_input, *d_weight, *d_bias, *d_output;
    hipMalloc(&d_input, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));
    hipMalloc(&d_weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float));
    hipMalloc(&d_bias, bias_dim0 * sizeof(float));
    hipMalloc(&d_output, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, weight_dim0 * weight_dim1 * weight_dim2 * weight_dim3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, bias_dim0 * sizeof(float), hipMemcpyHostToDevice);

    // Perform convolution
    hipdnnConvolutionForward(cudnnHandle,
                            &alpha,
                            inputDesc, d_input,
                            filterDesc, d_weight,
                            convDesc,
                            biasDesc, d_bias,
                            &beta,
                            outputDesc, d_output);

    // Apply ReLU activation
    hipdnnActivationDescriptor_t reluDesc;
    hipdnnCreateActivationDescriptor(&reluDesc);
    hipdnnSetActivationDescriptor(reluDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0f);
    hipdnnActivationForward(cudnnHandle,
                          reluDesc,
                          outputDesc, d_output,
                          outputDesc, d_output);

    // Copy result back to host
    hipMemcpy(output, d_output, input_tensor_dim0 * input_tensor_dim1 * input_tensor_dim2 * input_tensor_dim3 * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_output);

    // Destroy cudnn objects
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(weightDesc);
    hipdnnDestroyTensorDescriptor(biasDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroyActivationDescriptor(reluDesc);
    hipdnnDestroy(cudnnHandle);
}

}
