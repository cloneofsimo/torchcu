
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

#include <stdarg.h>

// Helper function to convert float to __nv_bfloat16
__device__ __forceinline__ __hip_bfloat16 float_to_bfloat16(float f) {
    return __float2bfloat16(f);
}

// Helper function to convert __nv_bfloat16 to float
__device__ __forceinline__ float bfloat16_to_float(__hip_bfloat16 bf) {
    return __bfloat162float(bf);
}

__global__ void pairwise_distance_bf16_kernel(const float* input1, const float* input2, float* output, 
                                            int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            __hip_bfloat16 a = float_to_bfloat16(input1[row * k + i]);
            __hip_bfloat16 b = float_to_bfloat16(input2[col * k + i]);
            sum += bfloat16_to_float(__hmul(a - b, a - b)); 
        }
        output[row * n + col] = sum;
    }
}

extern "C" {

void pairwise_distance_bf16(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensors
    const float* input1 = va_arg(args, const float*);
    int input1_dim0 = va_arg(args, int);
    int input1_dim1 = va_arg(args, int);

    const float* input2 = va_arg(args, const float*);
    int input2_dim0 = va_arg(args, int);
    int input2_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int m = input1_dim0;
    int n = input2_dim0;
    int k = input1_dim1;

    // Allocate device memory
    float *d_input1, *d_input2, *d_output;
    hipMalloc(&d_input1, m * k * sizeof(float));
    hipMalloc(&d_input2, n * k * sizeof(float));
    hipMalloc(&d_output, m * n * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input1, input1, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input2, input2, n * k * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    pairwise_distance_bf16_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input1, d_input2, d_output, m, n, k
    );

    // Copy result back to host
    hipMemcpy(output, d_output, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
}

}  // extern "C"
