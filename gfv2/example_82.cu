#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <>
#include <complex>
#include <math.h>
#include <stdarg.h> 

#define PI 3.14159265358979323846

// Helper functions for complex numbers
__device__ __forceinline__ std::complex<float> complex_mul(std::complex<float> a, std::complex<float> b) {
    return std::complex<float>(a.real() * b.real() - a.imag() * b.imag(), a.real() * b.imag() + a.imag() * b.real());
}

// Function to perform FFT on a 1D array (real-to-complex)
__device__ void fft1D(std::complex<float> *x, int n) {
    if (n == 1) return;

    std::complex<float> even[n / 2], odd[n / 2];
    for (int i = 0; i < n / 2; ++i) {
        even[i] = x[2 * i];
        odd[i] = x[2 * i + 1];
    }

    fft1D(even, n / 2);
    fft1D(odd, n / 2);

    for (int i = 0; i < n / 2; ++i) {
        float angle = -2 * PI * i / n;
        std::complex<float> factor(cos(angle), sin(angle));
        x[i] = even[i] + complex_mul(factor, odd[i]);
        x[i + n / 2] = even[i] - complex_mul(factor, odd[i]);
    }
}

// Function to perform inverse FFT on a 1D array (complex-to-real)
__device__ void ifft1D(std::complex<float> *x, int n) {
    // Conjugate the input array
    for (int i = 0; i < n; ++i) {
        x[i] = std::conj(x[i]);
    }

    // Perform FFT
    fft1D(x, n);

    // Conjugate again and divide by N
    for (int i = 0; i < n; ++i) {
        x[i] = std::conj(x[i]) / n;
    }
}

// CUDA kernel for int8 FFT convolution 1D
__global__ void int8_fft_conv1d_kernel(const int8_t *input, const int8_t *weight, const float *bias, float *output, 
                                         int batch_size, int in_channels, int out_channels, int input_size, 
                                         int kernel_size, int stride, int padding, int dilation, int groups, int mode) {
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int out_channel_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (batch_idx < batch_size && out_channel_idx < out_channels) {
        int out_idx = batch_idx * out_channels + out_channel_idx;

        // Padding
        int padding_start = (mode == 0) ? padding : 0; // Same padding
        int padding_end = (mode == 0) ? padding : padding * 2; // Same padding

        int input_size_padded = input_size + padding_start + padding_end;

        // Output size calculation
        int output_size = (input_size_padded - dilation * (kernel_size - 1) - 1) / stride + 1;

        // Allocate memory for FFT on device
        std::complex<float> *d_input_fft, *d_weight_fft, *d_output_fft;
        hipMalloc(&d_input_fft, input_size_padded * sizeof(std::complex<float>));
        hipMalloc(&d_weight_fft, kernel_size * sizeof(std::complex<float>));
        hipMalloc(&d_output_fft, output_size * sizeof(std::complex<float>));

        // Copy input to device
        hipMemcpy(d_input_fft, input + batch_idx * in_channels * input_size, input_size_padded * sizeof(float), hipMemcpyHostToDevice);

        // Zero-pad input for FFT
        for (int i = input_size; i < input_size_padded; ++i) {
            ((float *)d_input_fft)[i] = 0.0f;
        }

        // Copy weight to device
        hipMemcpy(d_weight_fft, weight + out_channel_idx * kernel_size, kernel_size * sizeof(float), hipMemcpyHostToDevice);

        // Perform FFT
        fft1D(d_input_fft, input_size_padded);
        fft1D(d_weight_fft, kernel_size);

        // Multiply in frequency domain
        for (int i = 0; i < output_size; ++i) {
            for (int k = 0; k < kernel_size; ++k) {
                int input_idx = i * stride + k * dilation; 

                if (input_idx >= 0 && input_idx < input_size_padded) {
                    d_output_fft[i] = complex_mul(d_output_fft[i], d_weight_fft[k]);
                    d_output_fft[i] = complex_mul(d_output_fft[i], d_input_fft[input_idx]);
                }
            }
        }

        // Inverse FFT
        ifft1D(d_output_fft, output_size);

        // Copy output from device
        hipMemcpy(output + out_idx, d_output_fft, output_size * sizeof(float), hipMemcpyDeviceToHost);

        // Apply bias
        output[out_idx] += bias[out_channel_idx];

        // Free device memory
        hipFree(d_input_fft);
        hipFree(d_weight_fft);
        hipFree(d_output_fft);
    }
}

extern "C" {

void int8_fft_conv1d(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const int8_t* input = va_arg(args, const int8_t*);
    int batch_size = va_arg(args, int);
    int in_channels = va_arg(args, int);
    int input_size = va_arg(args, int);

    // Extract weight tensor
    const int8_t* weight = va_arg(args, const int8_t*);
    int out_channels = va_arg(args, int);
    int kernel_size = va_arg(args, int);

    // Extract bias tensor
    const float* bias = va_arg(args, const float*);

    // Extract other arguments
    int stride = va_arg(args, int);
    int padding = va_arg(args, int);
    int dilation = va_arg(args, int);
    int groups = va_arg(args, int);
    int mode = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);

    va_end(args);

    // Calculate output size
    int output_size = (input_size + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

    // Launch kernel
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (out_channels + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int8_fft_conv1d_kernel<<<numBlocks, threadsPerBlock>>>(
        input, weight, bias, output, batch_size, in_channels, out_channels, input_size,
        kernel_size, stride, padding, dilation, groups, mode
    );
}

} // extern "C"
