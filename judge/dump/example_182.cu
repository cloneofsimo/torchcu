
#include <hip/hip_runtime.h>

#include <stdarg.h>  // Add this for va_list, va_start, va_end

extern "C" {

__global__ void adversarial_training_kernel(const float* input_tensor, const float* weights, const float* perturbation, 
                                            float* output, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += (input_tensor[row * k + i] + perturbation[row * k + i]) * weights[col * k + i];  // Transposed access
        }
        output[row * n + col] = sum;
    }
}

void adversarial_training_example(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);
    int input_tensor_dim1 = va_arg(args, int);

    // Extract weights tensor
    const float* weights = va_arg(args, const float*);
    int weights_dim0 = va_arg(args, int);
    int weights_dim1 = va_arg(args, int);

    // Extract perturbation tensor
    const float* perturbation = va_arg(args, const float*);
    int perturbation_dim0 = va_arg(args, int);
    int perturbation_dim1 = va_arg(args, int);

    // Extract output tensor (assuming it's preallocated)
    float* output = va_arg(args, float*);

    va_end(args);

    int batch_size = input_tensor_dim0;
    int input_dim = input_tensor_dim1;
    int output_dim = weights_dim0;

    // Allocate device memory
    float *d_input, *d_weights, *d_perturbation, *d_output;
    hipMalloc(&d_input, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_weights, output_dim * input_dim * sizeof(float));
    hipMalloc(&d_perturbation, batch_size * input_dim * sizeof(float));
    hipMalloc(&d_output, batch_size * output_dim * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, output_dim * input_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_perturbation, perturbation, batch_size * input_dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((output_dim + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (batch_size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    adversarial_training_kernel<<<numBlocks, threadsPerBlock>>>(
        d_input, d_weights, d_perturbation, d_output, batch_size, output_dim, input_dim
    );

    // Copy result back to host
    hipMemcpy(output, d_output, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_perturbation);
    hipFree(d_output);
}

}  // extern "C"
