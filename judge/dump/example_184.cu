
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdarg.h> 

// CUDA kernel for calculating the mean of a tensor using fp16
__global__ void mean_kernel_fp16(const float* input_tensor, float* output, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        // Load input as half precision
        __half input_half = __float2half_rn(input_tensor[i]);
        // Accumulate sum using half precision
        __half sum = __int2half_rn(0);
        sum = __hadd(sum, input_half);
        // Store result as float
        output[0] = __half2float(sum) / size;
    }
}

extern "C" {

void mean_fp16_function(int num_args, ...) {
    va_list args;
    va_start(args, num_args);

    // Extract input tensor
    const float* input_tensor = va_arg(args, const float*);
    int input_tensor_dim0 = va_arg(args, int);

    // Extract output tensor
    float* output = va_arg(args, float*);

    va_end(args);

    int size = input_tensor_dim0;

    // Allocate device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, input_tensor, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 threadsPerBlock(256);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    mean_kernel_fp16<<<numBlocks, threadsPerBlock>>>(d_input, d_output, size);

    // Copy result back to host
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

}  // extern "C"
